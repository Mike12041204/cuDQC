#include "hip/hip_runtime.h"
#include "../inc/common.h"
#include "../inc/device_kernels.h"

// --- PRIMARY KERNELS ---
__global__ void d_expand_level(GPU_Data* dd)
{
    __shared__ Warp_Data wd;        // data is stored in data structures to reduce the number of variables that need to be passed to methods
    Local_Data ld;
    int num_mem;                    // helper variables, not passed through to any methods
    int method_return;
    int index;

    // --- CURRENT LEVEL ---

    // reset warp tasks and cliques counts
    if (LANE_IDX == 0) {
        dd->wtasks_count[WARP_IDX] = 0;
        dd->wcliques_count[WARP_IDX] = 0;
    }
    __syncwarp();

    // initialize i for each warp
    int i = WARP_IDX;
    
    while (i < *dd->tasks_count) {

        // get information on vertices being handled within tasks
        if (LANE_IDX == 0) {
            wd.start[WIB_IDX] = dd->tasks_offset[i];
            wd.end[WIB_IDX] = dd->tasks_offset[i + 1];
            wd.tot_vert[WIB_IDX] = wd.end[WIB_IDX] - wd.start[WIB_IDX];
        }
        __syncwarp();

        // each warp gets partial number of members
        num_mem = 0;
        for (uint64_t j = wd.start[WIB_IDX] + LANE_IDX; j < wd.end[WIB_IDX]; j += WARP_SIZE) {
            if (dd->tasks_vertices[j].label != 1) {
                break;
            }
            num_mem++;
        }
        // sum members across warp
        for (int k = 1; k < 32; k *= 2) {
            num_mem += __shfl_xor_sync(0xFFFFFFFF, num_mem, k);
        }

        if (LANE_IDX == 0) {
            wd.num_mem[WIB_IDX] = num_mem;
            wd.num_cand[WIB_IDX] = wd.tot_vert[WIB_IDX] - wd.num_mem[WIB_IDX];
            wd.expansions[WIB_IDX] = wd.num_cand[WIB_IDX];
        }
        __syncwarp();

        // LOOKAHEAD PRUNING
        method_return = d_lookahead_pruning(dd, wd, ld);
        if (method_return) {
            // schedule warps next task
            if (LANE_IDX == 0) {
                i = atomicAdd(dd->current_task, 1);
            }
            i = __shfl_sync(0xFFFFFFFF, i, 0);
            continue;
        }

        // --- NEXT LEVEL ---
        for (int j = 0; j < wd.expansions[WIB_IDX]; j++)
        {

            // REMOVE ONE VERTEX
            if (j > 0) {
                method_return = d_remove_one_vertex(dd, wd, ld);
                if (method_return) {
                    break;
                }
            }

            // INITIALIZE NEW VERTICES
            if (LANE_IDX == 0) {
                wd.number_of_members[WIB_IDX] = wd.num_mem[WIB_IDX];
                wd.number_of_candidates[WIB_IDX] = wd.num_cand[WIB_IDX];
                wd.total_vertices[WIB_IDX] = wd.tot_vert[WIB_IDX];
            }
            __syncwarp();

            // select whether to store vertices in global or shared memory based on size
            if (wd.total_vertices[WIB_IDX] <= VERTICES_SIZE) {
                ld.vertices = wd.shared_vertices + (VERTICES_SIZE * WIB_IDX);
            }
            else {
                ld.vertices = dd->global_vertices + (*dd->wvertices_size * WARP_IDX);
            }

            // copy vertices
            for (index = LANE_IDX; index < wd.number_of_members[WIB_IDX]; index += WARP_SIZE) {
                ld.vertices[index] = dd->tasks_vertices[wd.start[WIB_IDX] + index];
            }
            for (; index < wd.total_vertices[WIB_IDX] - 1; index += WARP_SIZE) {
                ld.vertices[index + 1] = dd->tasks_vertices[wd.start[WIB_IDX] + index];
            }
            if (LANE_IDX == 0) {
                ld.vertices[wd.number_of_members[WIB_IDX]] = dd->tasks_vertices[wd.start[WIB_IDX] + wd.total_vertices[WIB_IDX] - 1];
            }
            __syncwarp();

            // ADD ONE VERTEX
            method_return = d_add_one_vertex(dd, wd, ld);

            // if failed found check for clique and continue on to the next iteration
            if (method_return == 1) {
                if (wd.number_of_members[WIB_IDX] >= (*dd->minimum_clique_size)) {
                    d_check_for_clique(dd, wd, ld);
                }
                continue;
            }

            // CRITICAL VERTEX PRUNING
            method_return = d_critical_vertex_pruning(dd, wd, ld);

            // critical fail, cannot be clique continue onto next iteration
            if (method_return == 2) {
                continue;
            }

            // HANDLE CLIQUES
            if (wd.number_of_members[WIB_IDX] >= (*dd->minimum_clique_size)) {
                d_check_for_clique(dd, wd, ld);
            }

            // if vertex in x found as not extendable continue to next iteration
            if (method_return == 1) {
                continue;
            }

            // WRITE TASKS TO BUFFERS
            // sort vertices in Quick efficient enumeration order before writing
            d_oe_sort_vert(ld.vertices, wd.total_vertices[WIB_IDX], d_comp_vert_Q);

            if (wd.number_of_candidates[WIB_IDX] > 0) {
                d_write_to_tasks(dd, wd, ld);
            }
        }

        // schedule warps next task
        if (LANE_IDX == 0) {
            i = atomicAdd(dd->current_task, 1);
        }
        i = __shfl_sync(0xFFFFFFFF, i, 0);
    }

    uint64_t tasks_count;
    uint64_t tasks_size;
    uint64_t cliques_count;
    uint64_t cliques_size;
    int target_warp;
    int temp1;
    int temp2;
    int temp3;
    int temp4;

    // ensure all warps in block are done so we can perform transfer scan operations
    __syncthreads();
    // each block has 32 warps and each warp has 32 lanes so we can load block data into warp 0 for processing
    if(WIB_IDX == 0){
        target_warp = (BLOCK_IDX * WARPS_PER_BLOCK) + LANE_IDX;

        // each lane gets a warps data
        tasks_count = dd->wtasks_count[target_warp];
        tasks_size = dd->wtasks_offset[target_warp * *dd->wtasks_offset_size + tasks_count];
        cliques_count = dd->wcliques_count[target_warp];
        cliques_size = dd->wcliques_offset[target_warp * *dd->wcliques_offset_size + cliques_count];

        // TODO - see if there is a way to do scan without temp variables
        // lanes perform scan across data
        for (int i = 1; i < WARP_SIZE; i *= 2) {
            temp1 = __shfl_up_sync(0xFFFFFFFF, tasks_count, i, WARP_SIZE);
            temp2 = __shfl_up_sync(0xFFFFFFFF, tasks_size, i, WARP_SIZE);
            temp3 = __shfl_up_sync(0xFFFFFFFF, cliques_count, i, WARP_SIZE);
            temp4 = __shfl_up_sync(0xFFFFFFFF, cliques_size, i, WARP_SIZE);

            if (LANE_IDX >= i) {
                tasks_count += temp1;
                tasks_size += temp2;
                cliques_count += temp3;
                cliques_size += temp4;
            }
            __syncwarp();
        }

        // lanes write to global scan arrays
        dd->scan_tasks_count[target_warp] = tasks_count;
        dd->scan_tasks_size[target_warp] = tasks_size;
        dd->scan_cliques_count[target_warp] = cliques_count;
        dd->scan_cliques_size[target_warp] = cliques_size;

        // last lane write block sum information
        if(LANE_IDX == WARP_SIZE - 1){
            dd->block_tasks_count[BLOCK_IDX] = tasks_count;
            dd->block_tasks_size[BLOCK_IDX] = tasks_size;
            dd->block_cliques_count[BLOCK_IDX] = cliques_count;
            dd->block_cliques_size[BLOCK_IDX] = cliques_size;
        }
    }

    if (LANE_IDX == 0) {
        // sum to find tasks count
        atomicAdd(dd->total_tasks, dd->wtasks_count[WARP_IDX]);
        atomicAdd(dd->total_cliques, dd->wcliques_count[WARP_IDX]);
    }

    if (IDX == 0) {
        *dd->buffer_offset_start = *dd->buffer_count + 1;
        *dd->buffer_start = dd->buffer_offset[*dd->buffer_count];
        *dd->cliques_offset_start = *dd->cliques_count + 1;
        *dd->cliques_start = dd->cliques_offset[*dd->cliques_count];
    }
}

__global__ void transfer_buffers(GPU_Data* dd, uint64_t* tasks_count, uint64_t* buffer_count, uint64_t* cliques_count)
{
    __shared__ uint64_t tasks_write[WARPS_PER_BLOCK];                   // important data used in transfer
    __shared__ uint64_t tasks_offset_write[WARPS_PER_BLOCK];
    __shared__ uint64_t cliques_write[WARPS_PER_BLOCK];
    __shared__ uint64_t cliques_offset_write[WARPS_PER_BLOCK];
    __shared__ int tasks_end;

    __shared__ int twarp;                                               // temporary information used in calculating important data
    __shared__ int toffsetwrite;
    __shared__ int twrite;

    uint64_t tw;
    uint64_t tow;
    uint64_t cw;
    uint64_t cow;

    __shared__ uint64_t block_tasks_count[NUM_OF_BLOCKS];
    __shared__ uint64_t block_tasks_size[NUM_OF_BLOCKS];
    __shared__ uint64_t block_cliques_count[NUM_OF_BLOCKS];
    __shared__ uint64_t block_cliques_size[NUM_OF_BLOCKS];
    int partner;
    uint64_t prev;
    uint64_t curr;
    __shared__ int block_end;
    __shared__ int expand_diff;
    bool larger;
    uint32_t mask;
    int warp_end;
    int inter_end;

    uint64_t btw;
    uint64_t btow;
    uint64_t bcw;
    uint64_t bcow;

    // NEW WRITE CALCULATIONS

    // updated transfer buffers scan to get each warps write locations and tasks end
    // threads in block transfer scan information from global to shared memory
    for(int i = THREAD_IDX; i < NUM_OF_BLOCKS; i += BLOCK_SIZE){
        block_tasks_count[i] = dd->block_tasks_count[i];
        block_tasks_size[i] = dd->block_tasks_size[i];
        block_cliques_count[i] = dd->block_cliques_count[i];
        block_cliques_size[i] = dd->block_cliques_size[i];
    }
    __syncthreads();

    // perform inclusive scan operation
    for(int i = 1; i < BLOCK_SIZE; i *= 2){
        partner = THREAD_IDX - i;

        // if partner value is in valid range
        if(partner >= i && THREAD_IDX < NUM_OF_BLOCKS){
            block_tasks_count[THREAD_IDX] += block_tasks_count[partner];
            block_tasks_size[THREAD_IDX] += block_tasks_size[partner];
            block_cliques_count[THREAD_IDX] += block_cliques_count[partner];
            block_cliques_size[THREAD_IDX] += block_cliques_size[partner];
        }
        __syncthreads();
    }

    // use scan data to get tasks end
    // handle case where all data fits into tasks list
    if(block_tasks_count[NUM_OF_BLOCKS] <= *dd->expand_threshold){
        tasks_end = block_tasks_size[NUM_OF_BLOCKS];
    }
    else{
        // each thread detects whether it represents the block which surpases the expand threshold
        prev = 0;
        curr = 0;
        if(THREAD_IDX > 0){
            prev = block_tasks_count[THREAD_IDX - 1];
        }
        if(THREAD_IDX < NUM_OF_BLOCKS){
            curr = block_tasks_count[THREAD_IDX];
        }

        if(prev < *dd->expand_threshold && curr >= *dd->expand_threshold){
            block_end = THREAD_IDX;
            expand_diff = *dd->expand_threshold - prev;
        }
        __syncthreads();

        // we know now which block surpases the expand thrshold, from here we find the exact warp / task
        // since there are 32 warps per block we have narrowed the option to 32, thus we can fit the data in one warp and operate from there
        if(WARP_IDX == 0){
            // find first thread to have value larger than or equal to expand diff
            larger = false;
            if(dd->scan_tasks_count[(WARPS_PER_BLOCK * block_end) + LANE_IDX] >= expand_diff){
                larger = true;
            }

            mask = __ballot_sync(0xFFFFFFFF, larger);
            warp_end = __ffs(mask) - 1; // __ffs returns 1-based index, so subtract 1

            // we have found the warp where we exceed the expand threshold, find which task in the warp it is
            if(LANE_IDX == 0){
                inter_end = dd->wtasks_count[(WARPS_PER_BLOCK * block_end) + warp_end] - (dd->scan_tasks_count[(WARPS_PER_BLOCK * block_end) + warp_end] - expand_diff);

                // we now know exactly which tasks exceeds the expand threshold, so get the size for all the tasks which will go in the tasks list
                prev = 0;
                if(block_end > 0){
                    prev = block_tasks_size[block_end - 1];
                }

                tasks_end = prev + dd->wtasks_offset[(*dd->wtasks_offset_size * ((WARPS_PER_BLOCK * block_end) + warp_end - 1)) + inter_end];
            }
        }
    }

    // tasks end found now find write locaitons for every warp
    if(LANE_IDX == 0){
        btw = 0;
        btow = 0;
        bcw = 0;
        bcow = 0;
        // get each warps block offset
        if(BLOCK_IDX > 0){
            btow = block_tasks_count[BLOCK_IDX - 1];
            btw = block_tasks_size[BLOCK_IDX - 1];
            bcow = block_cliques_count[BLOCK_IDX - 1];
            bcw = block_cliques_size[BLOCK_IDX - 1];
        }

        tw = 0;
        tow = 0;
        cw = 0;
        cow = 0;
        // get each warps offset
        if(WARP_IDX % WARPS_PER_BLOCK > 0){
            tow = dd->scan_tasks_count[WARP_IDX - 1];
            tw = dd->scan_tasks_size[WARP_IDX - 1];
            cow = dd->scan_cliques_count[WARP_IDX - 1];
            cw = dd->scan_cliques_size[WARP_IDX - 1];
        }

        tasks_write[WIB_IDX] = btw + tw;
        tasks_offset_write[WIB_IDX] = 1 + btow + tow;
        cliques_write[WIB_IDX] = bcw + cw;
        cliques_offset_write[WIB_IDX] = 1 + bcow + cow;
    }
    __syncwarp();

    // OLD WRITE CALCULATIONS

    // point of this is to find how many vertices will be transfered to tasks, it is easy to know how many tasks as it will just
    // be the expansion threshold, but to find how many vertices we must now the total size of all the tasks that will be copied.
    // each block does this but really could be done by one thread outside the GPU
    if (THREAD_IDX == 0) {
        twarp = -1;
        toffsetwrite = 0;
        twrite = 0;

        for (int i = 0; i < NUMBER_OF_WARPS; i++) {
            // if next warps count is more than expand threshold mark as such and break
            if (toffsetwrite + dd->wtasks_count[i] >= *dd->expand_threshold) {
                twarp = i;
                break;
            }
            // else adds its size and count
            twrite += dd->wtasks_offset[(*dd->wtasks_offset_size * i) + dd->wtasks_count[i]];
            toffsetwrite += dd->wtasks_count[i];
        }
        // final size is the size of all tasks up until last warp and the remaining tasks in the last warp until expand threshold is satisfied
        tasks_end = twrite;
        if(twarp != -1){
            tasks_end += dd->wtasks_offset[(*dd->wtasks_offset_size * twarp) + (*dd->expand_threshold - toffsetwrite)];
        }
    }
    __syncthreads();

    // get each warps offsets for tasks and cliques by having eahc lane get partial and then summing
    tw = 0;
    tow = 0;
    cw = 0;
    cow = 0;
    for (int i = LANE_IDX; i < WARP_IDX; i += WARP_SIZE) {
        tow += dd->wtasks_count[i];
        tw += dd->wtasks_offset[(*dd->wtasks_offset_size * i) + dd->wtasks_count[i]];

        cow += dd->wcliques_count[i];
        cw += dd->wcliques_offset[(*dd->wcliques_offset_size * i) + dd->wcliques_count[i]];
    }

    // get sum
    for (int i = 1; i < WARP_SIZE; i *= 2) {
        tw += __shfl_xor_sync(0xFFFFFFFF, tw, i);
        tow += __shfl_xor_sync(0xFFFFFFFF, tow, i);
        cw += __shfl_xor_sync(0xFFFFFFFF, cw, i);
        cow += __shfl_xor_sync(0xFFFFFFFF, cow, i);
    }

    // warp level
    if (LANE_IDX == 0) {
        tasks_write[WIB_IDX] = tw;
        tasks_offset_write[WIB_IDX] = 1 + tow;
        cliques_write[WIB_IDX] = cw;
        cliques_offset_write[WIB_IDX] = 1 + cow;
    }
    __syncwarp();
    
    // move to tasks and buffer
    for (int i = LANE_IDX + 1; i <= dd->wtasks_count[WARP_IDX]; i += WARP_SIZE) {
        if (tasks_offset_write[WIB_IDX] + i - 1 <= *dd->expand_threshold) {
            // to tasks
            dd->tasks_offset[tasks_offset_write[WIB_IDX] + i - 1] = dd->wtasks_offset[(*dd->wtasks_offset_size * WARP_IDX) + i] + tasks_write[WIB_IDX];
        }
        else {
            // to buffer
            dd->buffer_offset[tasks_offset_write[WIB_IDX] + i - 2 - *dd->expand_threshold + *dd->buffer_offset_start] = dd->wtasks_offset[(*dd->wtasks_offset_size * WARP_IDX) + i] +
                tasks_write[WIB_IDX] - tasks_end + *dd->buffer_start;
        }
    }

    for (int i = LANE_IDX; i < dd->wtasks_offset[(*dd->wtasks_offset_size * WARP_IDX) + dd->wtasks_count[WARP_IDX]]; i += WARP_SIZE) {
        if (tasks_write[WIB_IDX] + i < tasks_end) {
            // to tasks
            dd->tasks_vertices[tasks_write[WIB_IDX] + i] = dd->wtasks_vertices[(*dd->wtasks_size * WARP_IDX) + i];
        }
        else {
            // to buffer
            dd->buffer_vertices[*dd->buffer_start + tasks_write[WIB_IDX] + i - tasks_end] = dd->wtasks_vertices[(*dd->wtasks_size * WARP_IDX) + i];
        }
    }
    // NOTE - this sync is important for some reason, larger graphs/et dont work without it
    __syncthreads();

    //move to cliques
    for (int i = LANE_IDX + 1; i <= dd->wcliques_count[WARP_IDX]; i += WARP_SIZE) {
        dd->cliques_offset[*dd->cliques_offset_start + cliques_offset_write[WIB_IDX] + i - 2] = dd->wcliques_offset[(*dd->wcliques_offset_size * WARP_IDX) + i] + *dd->cliques_start + 
            cliques_write[WIB_IDX];
    }
    for (int i = LANE_IDX; i < dd->wcliques_offset[(*dd->wcliques_offset_size * WARP_IDX) + dd->wcliques_count[WARP_IDX]]; i += WARP_SIZE) {
        dd->cliques_vertex[*dd->cliques_start + cliques_write[WIB_IDX] + i] = dd->wcliques_vertex[(*dd->wcliques_size * WARP_IDX) + i];
    }

    // reset some values for the next round within the kernel to prevent device synchronization
    if (IDX == 0) {
        // handle tasks and buffer counts
        if (*dd->total_tasks <= *dd->expand_threshold) {
            *dd->tasks_count = *dd->total_tasks;
        }
        else {
            *dd->tasks_count = *dd->expand_threshold;
            *dd->buffer_count += *dd->total_tasks - *dd->expand_threshold;
        }
        *dd->cliques_count += *dd->total_cliques;

        *dd->total_tasks = 0;
        *dd->total_cliques = 0;
        (*dd->current_level)++;

        *dd->current_task = NUMBER_OF_WARPS;
        *tasks_count = *dd->tasks_count;
        *buffer_count = *dd->buffer_count;
        *cliques_count = *dd->cliques_count;
    }
}

__global__ void fill_from_buffer(GPU_Data* dd, uint64_t* buffer_count)
{
    // get read and write locations
    int write_amount = (*dd->buffer_count >= *dd->expand_threshold - *dd->tasks_count) ? *dd->expand_threshold - *dd->tasks_count : *dd->buffer_count;
    uint64_t start_buffer = dd->buffer_offset[*dd->buffer_count - write_amount];
    uint64_t end_buffer = dd->buffer_offset[*dd->buffer_count];
    uint64_t size_buffer = end_buffer - start_buffer;
    uint64_t start_write = dd->tasks_offset[*dd->tasks_count];

    // handle offsets
    for (int i = IDX + 1; i <= write_amount; i += NUMBER_OF_THREADS) {
        dd->tasks_offset[*dd->tasks_count + i] = start_write + dd->buffer_offset[*dd->buffer_count - write_amount + i] - start_buffer;
    }

    // handle data
    for (int i = IDX; i < size_buffer; i += NUMBER_OF_THREADS) {
        dd->tasks_vertices[start_write + i] = dd->buffer_vertices[start_buffer + i];
    }

    if (IDX == 0) {
        *dd->tasks_count += write_amount;
        *dd->buffer_count -= write_amount;

        *buffer_count = *dd->buffer_count;
    }
}

// --- SECONDARY EXPANSION KERNELS ---
// returns 1 if lookahead succesful, 0 otherwise  
__device__ int d_lookahead_pruning(GPU_Data* dd, Warp_Data& wd, Local_Data& ld)
{
    int pvertexid;
    int phelper1;
    int phelper2;
    uint64_t start_write;

    if (LANE_IDX == 0) {
        wd.success[WIB_IDX] = true;
    }
    __syncwarp();

    // check if members meet degree requirement, dont need to check 2hop adj as diameter pruning guarentees all members will be within 2hops of eveything
    for (int i = LANE_IDX; i < wd.num_mem[WIB_IDX] && wd.success[WIB_IDX]; i += WARP_SIZE) {
        if (dd->tasks_vertices[wd.start[WIB_IDX] + i].indeg + dd->tasks_vertices[wd.start[WIB_IDX] + i].exdeg < dd->minimum_degrees[wd.tot_vert[WIB_IDX]]) {
            wd.success[WIB_IDX] = false;
            break;
        }
    }
    __syncwarp();

    if (!wd.success[WIB_IDX]) {
        return 0;
    }

    // update lvl2adj to candidates for all vertices
    for (int i = wd.num_mem[WIB_IDX] + LANE_IDX; i < wd.tot_vert[WIB_IDX]; i += WARP_SIZE) {
        pvertexid = dd->tasks_vertices[wd.start[WIB_IDX] + i].vertexid;
        
        for (int j = wd.num_mem[WIB_IDX]; j < wd.tot_vert[WIB_IDX]; j++) {
            if (j == i) {
                continue;
            }

            phelper1 = dd->tasks_vertices[wd.start[WIB_IDX] + j].vertexid;
            phelper2 = d_b_search_int(dd->twohop_neighbors + dd->twohop_offsets[phelper1], dd->twohop_offsets[phelper1 + 1] - dd->twohop_offsets[phelper1], pvertexid);
        
            if (phelper2 > -1) {
                dd->tasks_vertices[wd.start[WIB_IDX] + i].lvl2adj++;
            }
        }
    }
    __syncwarp();

    // compares all vertices to the lemmas from Quick
    for (int j = wd.num_mem[WIB_IDX] + LANE_IDX; j < wd.tot_vert[WIB_IDX] && wd.success[WIB_IDX]; j += WARP_SIZE) {
        if (dd->tasks_vertices[wd.start[WIB_IDX] + j].lvl2adj < wd.num_cand[WIB_IDX] - 1 || dd->tasks_vertices[wd.start[WIB_IDX] + j].indeg + dd->tasks_vertices[wd.start[WIB_IDX] + j].exdeg < dd->minimum_degrees[wd.tot_vert[WIB_IDX]]) {
            wd.success[WIB_IDX] = false;
            break;
        }
    }
    __syncwarp();

    if (wd.success[WIB_IDX]) {
        // write to cliques
        start_write = (*dd->wcliques_size * WARP_IDX) + dd->wcliques_offset[(*dd->wcliques_offset_size * WARP_IDX) + dd->wcliques_count[WARP_IDX]];
        for (int j = LANE_IDX; j < wd.tot_vert[WIB_IDX]; j += WARP_SIZE) {
            dd->wcliques_vertex[start_write + j] = dd->tasks_vertices[wd.start[WIB_IDX] + j].vertexid;
        }
        if (LANE_IDX == 0) {
            (dd->wcliques_count[WARP_IDX])++;
            dd->wcliques_offset[(*dd->wcliques_offset_size * WARP_IDX) + dd->wcliques_count[WARP_IDX]] = start_write - (*dd->wcliques_size * WARP_IDX) + wd.tot_vert[WIB_IDX];
        }
        return 1;
    }

    return 0;
}

// returns 1 if failed found after removing, 0 otherwise
__device__ int d_remove_one_vertex(GPU_Data* dd, Warp_Data& wd, Local_Data& ld)
{
    int pvertexid;
    int phelper1;
    int phelper2;
    int mindeg;

    mindeg = d_get_mindeg(wd.num_mem[WIB_IDX], dd);

    // remove the last candidate in vertices
    if (LANE_IDX == 0) {
        wd.num_cand[WIB_IDX]--;
        wd.tot_vert[WIB_IDX]--;
        wd.success[WIB_IDX] = false;
    }
    __syncwarp();

    // update info of vertices connected to removed cand
    pvertexid = dd->tasks_vertices[wd.start[WIB_IDX] + wd.tot_vert[WIB_IDX]].vertexid;

    for (int i = LANE_IDX; i < wd.tot_vert[WIB_IDX] && !wd.success[WIB_IDX]; i += WARP_SIZE) {
        phelper1 = dd->tasks_vertices[wd.start[WIB_IDX] + i].vertexid;
        phelper2 = d_b_search_int(dd->onehop_neighbors + dd->onehop_offsets[pvertexid], dd->onehop_offsets[pvertexid + 1] - dd->onehop_offsets[pvertexid], phelper1);

        if (phelper2 > -1) {
            dd->tasks_vertices[wd.start[WIB_IDX] + i].exdeg--;

            if (phelper1 < wd.num_mem[WIB_IDX] && dd->tasks_vertices[wd.start[WIB_IDX] + phelper1].indeg + dd->tasks_vertices[wd.start[WIB_IDX] + phelper1].exdeg < mindeg) {
                wd.success[WIB_IDX] = true;
                break;
            }
        }
    }
    __syncwarp();

    if (wd.success[WIB_IDX]) {
        return 1;
    }

    return 0;
}

// returns 1 if failed found or invalid bound, 0 otherwise 
__device__ int d_add_one_vertex(GPU_Data* dd, Warp_Data& wd, Local_Data& ld)
{
    int pvertexid;
    int phelper1;
    int phelper2;
    bool failed_found;

    // ADD ONE VERTEX
    pvertexid = ld.vertices[wd.number_of_members[WIB_IDX]].vertexid;

    if (LANE_IDX == 0) {
        ld.vertices[wd.number_of_members[WIB_IDX]].label = 1;
        wd.number_of_members[WIB_IDX]++;
        wd.number_of_candidates[WIB_IDX]--;
    }
    __syncwarp();

    for (int i = LANE_IDX; i < wd.tot_vert[WIB_IDX]; i += WARP_SIZE) {
        phelper1 = ld.vertices[i].vertexid;
        phelper2 = d_b_search_int(dd->onehop_neighbors + dd->onehop_offsets[pvertexid], dd->onehop_offsets[pvertexid + 1] - dd->onehop_offsets[pvertexid], phelper1);

        if (phelper2 > -1) {
            ld.vertices[i].exdeg--;
            ld.vertices[i].indeg++;
        }
    }
    __syncwarp();

    // DIAMETER PRUNING
    d_diameter_pruning(dd, wd, ld, pvertexid);

    // DEGREE BASED PRUNING
    failed_found = d_degree_pruning(dd, wd, ld);

    // if vertex in x found as not extendable continue to next iteration
    if (failed_found) {
        return 1;
    }
   
    return 0;
}

// returns 2, if critical fail, 1 if failed found or invalid bound, 0 otherwise
__device__ int d_critical_vertex_pruning(GPU_Data* dd, Warp_Data& wd, Local_Data& ld)
{
    int phelper1;                   // intersection
    int number_of_crit_adj;         // pruning
    bool failed_found;

    // CRITICAL VERTEX PRUNING 
    // iterate through all vertices in clique
    for (int k = 0; k < wd.number_of_members[WIB_IDX]; k++) {

        // if they are a critical vertex
        if (ld.vertices[k].indeg + ld.vertices[k].exdeg == dd->minimum_degrees[wd.number_of_members[WIB_IDX] + wd.lower_bound[WIB_IDX]] && ld.vertices[k].exdeg > 0) {
            phelper1 = ld.vertices[k].vertexid;

            // iterate through all candidates
            for (int i = wd.number_of_members[WIB_IDX] + LANE_IDX; i < wd.total_vertices[WIB_IDX]; i += WARP_SIZE) {
                if (ld.vertices[i].label != 4) {
                    // if candidate is neighbor of critical vertex mark as such
                    if (d_b_search_int(dd->onehop_neighbors + dd->onehop_offsets[phelper1], dd->onehop_offsets[phelper1 + 1] - dd->onehop_offsets[phelper1], ld.vertices[i].vertexid) > -1) {
                        ld.vertices[i].label = 4;
                    }
                }
            }
        }
        __syncwarp();
    }

    // sort vertices so that critical vertex adjacent candidates are immediately after vertices within the clique
    d_oe_sort_vert(ld.vertices + wd.number_of_members[WIB_IDX], wd.number_of_candidates[WIB_IDX], d_comp_vert_cv);

    // count number of critical adjacent vertices
    number_of_crit_adj = 0;
    for (int i = wd.number_of_members[WIB_IDX] + LANE_IDX; i < wd.total_vertices[WIB_IDX]; i += WARP_SIZE) {
        if (ld.vertices[i].label == 4) {
            number_of_crit_adj++;
        }
        else {
            break;
        }
    }
    // get sum
    for (int i = 1; i < WARP_SIZE; i *= 2) {
        number_of_crit_adj += __shfl_xor_sync(0xFFFFFFFF, number_of_crit_adj, i);
    }

    failed_found = false;

    // reset adjacencies
    for (int i = LANE_IDX; i < wd.total_vertices[WIB_IDX]; i += WARP_SIZE) {
        dd->adjacencies[(*dd->wvertices_size * WARP_IDX) + i] = 0;
    }

    // if there were any neighbors of critical vertices
    if (number_of_crit_adj > 0)
    {
        // iterate through all vertices and update their degrees as if critical adjacencies were added and keep track of how many critical adjacencies they are adjacent to
        for (int k = LANE_IDX; k < wd.total_vertices[WIB_IDX]; k += WARP_SIZE) {
            phelper1 = ld.vertices[k].vertexid;

            for (int i = wd.number_of_members[WIB_IDX]; i < wd.number_of_members[WIB_IDX] + number_of_crit_adj; i++) {
                if (d_b_search_int(dd->onehop_neighbors + dd->onehop_offsets[phelper1], dd->onehop_offsets[phelper1 + 1] - dd->onehop_offsets[phelper1], ld.vertices[i].vertexid) > -1) {
                    ld.vertices[k].indeg++;
                    ld.vertices[k].exdeg--;
                }

                if (d_b_search_int(dd->twohop_neighbors + dd->twohop_offsets[phelper1], dd->twohop_offsets[phelper1 + 1] - dd->twohop_offsets[phelper1], ld.vertices[i].vertexid) > -1) {
                    dd->adjacencies[(*dd->wvertices_size * WARP_IDX) + k]++;
                }
            }
        }
        __syncwarp();

        // all vertices within the clique must be within 2hops of the newly added critical vertex adj vertices
        for (int k = LANE_IDX; k < wd.number_of_members[WIB_IDX]; k += WARP_SIZE) {
            if (dd->adjacencies[(*dd->wvertices_size * WARP_IDX) + k] != number_of_crit_adj) {
                failed_found = true;
                break;
            }
        }
        failed_found = __any_sync(0xFFFFFFFF, failed_found);
        if (failed_found) {
            return 2;
        }

        // all critical adj vertices must all be within 2 hops of each other
        for (int k = wd.number_of_members[WIB_IDX] + LANE_IDX; k < wd.number_of_members[WIB_IDX] + number_of_crit_adj; k += WARP_SIZE) {
            if (dd->adjacencies[(*dd->wvertices_size * WARP_IDX) + k] < number_of_crit_adj - 1) {
                failed_found = true;
                break;
            }
        }
        failed_found = __any_sync(0xFFFFFFFF, failed_found);
        if (failed_found) {
            return 2;
        }

        // no failed vertices found so add all critical vertex adj candidates to clique
        for (int k = wd.number_of_members[WIB_IDX] + LANE_IDX; k < wd.number_of_members[WIB_IDX] + number_of_crit_adj; k += WARP_SIZE) {
            ld.vertices[k].label = 1;
        }

        if (LANE_IDX == 0) {
            wd.number_of_members[WIB_IDX] += number_of_crit_adj;
            wd.number_of_candidates[WIB_IDX] -= number_of_crit_adj;
        }
        __syncwarp();
    }

    // DIAMTER PRUNING
    d_diameter_pruning_cv(dd, wd, ld, number_of_crit_adj);

    // DEGREE BASED PRUNING
    failed_found = d_degree_pruning(dd, wd, ld);

    // if vertex in x found as not extendable continue to next iteration
    if (failed_found) {
        return 1;
    }

    return 0;
}

// diameter pruning intitializes vertices labels and candidate indegs array for use in iterative degree pruning
__device__ void d_diameter_pruning(GPU_Data* dd, Warp_Data& wd, Local_Data& ld, int pvertexid)
{
    int lane_write;
    int phelper1;                       // intersection
    int phelper2;
    int lane_remaining_count;           // vertex iteration

    lane_write = (*dd->wvertices_size * WARP_IDX) + ((*dd->wvertices_size / WARP_SIZE) * LANE_IDX);
    lane_remaining_count = 0;

    for (int i = wd.number_of_members[WIB_IDX] + LANE_IDX; i < wd.total_vertices[WIB_IDX]; i += WARP_SIZE) {
        ld.vertices[i].label = -1;
    }
    __syncwarp();

    for (int i = wd.number_of_members[WIB_IDX] + LANE_IDX; i < wd.total_vertices[WIB_IDX]; i += WARP_SIZE) {
        phelper1 = ld.vertices[i].vertexid;
        phelper2 = d_b_search_int(dd->twohop_neighbors + dd->twohop_offsets[pvertexid], dd->twohop_offsets[pvertexid + 1] - dd->twohop_offsets[pvertexid], phelper1);

        if (phelper2 > -1) {
            ld.vertices[i].label = 0;
            dd->lane_candidate_indegs[lane_write + lane_remaining_count++] = ld.vertices[i].indeg;
        }
    }
    __syncwarp();

    // scan to calculate write postion in warp arrays
    phelper2 = lane_remaining_count;
    for (int i = 1; i < WARP_SIZE; i *= 2) {
        phelper1 = __shfl_up_sync(0xFFFFFFFF, lane_remaining_count, i, WARP_SIZE);
        if (LANE_IDX >= i) {
            lane_remaining_count += phelper1;
        }
        __syncwarp();
    }
    // lane remaining count sum is scan for last lane and its value
    if (LANE_IDX == WARP_SIZE - 1) {
        wd.remaining_count[WIB_IDX] = lane_remaining_count;
    }
    // make scan exclusive
    lane_remaining_count -= phelper2;
    __syncwarp();

    // parallel write lane arrays to warp array
    for (int i = 0; i < phelper2; i++) {
        dd->candidate_indegs[(*dd->wvertices_size * WARP_IDX) + lane_remaining_count + i] = dd->lane_candidate_indegs[lane_write + i];
    }
    __syncwarp();
}

__device__ void d_diameter_pruning_cv(GPU_Data* dd, Warp_Data& wd, Local_Data& ld, int number_of_crit_adj)
{
    int lane_write;
    int lane_remaining_count;           // vertex iteration
    int phelper1;                       // intersection
    int phelper2;

    lane_write = (*dd->wvertices_size * WARP_IDX) + ((*dd->wvertices_size / WARP_SIZE) * LANE_IDX);
    lane_remaining_count = 0;

    // remove all cands who are not within 2hops of all newly added cands
    for (int k = wd.number_of_members[WIB_IDX] + LANE_IDX; k < wd.total_vertices[WIB_IDX]; k += WARP_SIZE) {
        if (dd->adjacencies[(*dd->wvertices_size * WARP_IDX) + k] == number_of_crit_adj) {
            dd->lane_candidate_indegs[lane_write + lane_remaining_count++] = ld.vertices[k].indeg;
        }
        else {
            ld.vertices[k].label = -1;
        }
    }

    // scan to calculate write postion in warp arrays
    phelper2 = lane_remaining_count;
    for (int i = 1; i < WARP_SIZE; i *= 2) {
        phelper1 = __shfl_up_sync(0xFFFFFFFF, lane_remaining_count, i, WARP_SIZE);
        if (LANE_IDX >= i) {
            lane_remaining_count += phelper1;
        }
        __syncwarp();
    }
    // lane remaining count sum is scan for last lane and its value
    if (LANE_IDX == WARP_SIZE - 1) {
        wd.remaining_count[WIB_IDX] = lane_remaining_count;
    }
    // make scan exclusive
    lane_remaining_count -= phelper2;
    __syncwarp();

    // parallel write lane arrays to warp array
    for (int i = 0; i < phelper2; i++) {
        dd->candidate_indegs[(*dd->wvertices_size * WARP_IDX) + lane_remaining_count + i] = dd->lane_candidate_indegs[lane_write + i];
    }
    __syncwarp();
}

// returns true if invalid bounds or failed found
__device__ bool d_degree_pruning(GPU_Data* dd, Warp_Data& wd, Local_Data& ld)
{
    // vertices size * warp idx + (vertices size / warp size) * lane idx
    int lane_write = ((*dd->wvertices_size * WARP_IDX) + ((*dd->wvertices_size / WARP_SIZE) * LANE_IDX));

    // helper variables used throughout method to store various values, names have no meaning
    int pvertexid;
    int phelper1;
    int phelper2;
    Vertex* read;
    Vertex* write;
    // counter for lane intersection results
    int lane_remaining_count;
    int lane_removed_count;

    d_oe_sort_int(dd->candidate_indegs + (*dd->wvertices_size * WARP_IDX), wd.remaining_count[WIB_IDX], d_comp_int_desc);

    d_calculate_LU_bounds(dd, wd, ld, wd.remaining_count[WIB_IDX]);
    if (wd.success[WIB_IDX]) {
        return true;
    }

    // check for failed vertices
    __syncwarp();
    for (int k = LANE_IDX; k < wd.number_of_members[WIB_IDX] && !wd.success[WIB_IDX]; k += WARP_SIZE) {
        if (!d_vert_isextendable(ld.vertices[k], dd, wd, ld)) {
            wd.success[WIB_IDX] = true;
            break;
        }

    }
    __syncwarp();
    if (wd.success[WIB_IDX]) {
        return true;
    }

    if (LANE_IDX == 0) {
        wd.remaining_count[WIB_IDX] = 0;
        wd.removed_count[WIB_IDX] = 0;
        wd.rw_counter[WIB_IDX] = 0;
    }

    lane_remaining_count = 0;
    lane_removed_count = 0;
    
    for (int i = wd.number_of_members[WIB_IDX] + LANE_IDX; i < wd.total_vertices[WIB_IDX]; i += WARP_SIZE) {
        if (ld.vertices[i].label == 0 && d_cand_isvalid(ld.vertices[i], dd, wd, ld)) {
            dd->lane_remaining_candidates[lane_write + lane_remaining_count++] = i;
        }
        else {
            dd->lane_removed_candidates[lane_write + lane_removed_count++] = i;
        }
    }
    __syncwarp();

    // scan to calculate write postion in warp arrays
    phelper2 = lane_remaining_count;
    pvertexid = lane_removed_count;
    for (int i = 1; i < WARP_SIZE; i *= 2) {
        phelper1 = __shfl_up_sync(0xFFFFFFFF, lane_remaining_count, i, WARP_SIZE);
        if (LANE_IDX >= i) {
            lane_remaining_count += phelper1;
        }
        phelper1 = __shfl_up_sync(0xFFFFFFFF, lane_removed_count, i, WARP_SIZE);
        if (LANE_IDX >= i) {
            lane_removed_count += phelper1;
        }
        __syncwarp();
    }
    // lane remaining count sum is scan for last lane and its value
    if (LANE_IDX == WARP_SIZE - 1) {
        wd.remaining_count[WIB_IDX] = lane_remaining_count;
        wd.removed_count[WIB_IDX] = lane_removed_count;
    }
    // make scan exclusive
    lane_remaining_count -= phelper2;
    lane_removed_count -= pvertexid;

    // parallel write lane arrays to warp array
    for (int i = 0; i < phelper2; i++) {
        dd->remaining_candidates[(*dd->wvertices_size * WARP_IDX) + lane_remaining_count + i] = ld.vertices[dd->lane_remaining_candidates[lane_write + i]];
    }
    // only need removed if going to be using removed to update degrees
    if (!(wd.remaining_count[WIB_IDX] < wd.removed_count[WIB_IDX])) {
        for (int i = 0; i < pvertexid; i++) {
            dd->removed_candidates[(*dd->wvertices_size * WARP_IDX) + lane_removed_count + i] = ld.vertices[dd->lane_removed_candidates[lane_write + i]].vertexid;
        }
    }
    __syncwarp();
    
    while (wd.remaining_count[WIB_IDX] > 0 && wd.removed_count[WIB_IDX] > 0) {
        // we alternate reading and writing remaining variables from two arrays
        if (wd.rw_counter[WIB_IDX] % 2 == 0) {
            read = dd->remaining_candidates + (*dd->wvertices_size * WARP_IDX);
            write = ld.vertices + wd.number_of_members[WIB_IDX];
        }
        else {
            read = ld.vertices + wd.number_of_members[WIB_IDX];
            write = dd->remaining_candidates + (*dd->wvertices_size * WARP_IDX);
        }

        // update degrees
        if (wd.remaining_count[WIB_IDX] < wd.removed_count[WIB_IDX]) {
            // via remaining, reset exdegs
            for (int i = LANE_IDX; i < wd.number_of_members[WIB_IDX]; i += WARP_SIZE) {
                ld.vertices[i].exdeg = 0;
            }
            for (int i = LANE_IDX; i < wd.remaining_count[WIB_IDX]; i += WARP_SIZE) {
                read[i].exdeg = 0;
            }
            __syncwarp();

            // update exdeg based on remaining candidates, every lane should get the next vertex to intersect dynamically
            for (int i = LANE_IDX; i < wd.number_of_members[WIB_IDX]; i += WARP_SIZE) {
                pvertexid = ld.vertices[i].vertexid;

                for (int j = 0; j < wd.remaining_count[WIB_IDX]; j++) {
                    phelper1 = read[j].vertexid;
                    phelper2 = d_b_search_int(dd->onehop_neighbors + dd->onehop_offsets[phelper1], dd->onehop_offsets[phelper1 + 1] - dd->onehop_offsets[phelper1], pvertexid);

                    if (phelper2 > -1) {
                        ld.vertices[i].exdeg++;
                    }
                }
            }

            for (int i = LANE_IDX; i < wd.remaining_count[WIB_IDX]; i += WARP_SIZE) {
                pvertexid = read[i].vertexid;

                for (int j = 0; j < wd.remaining_count[WIB_IDX]; j++) {
                    if (j == i) {
                        continue;
                    }

                    phelper1 = read[j].vertexid;
                    phelper2 = d_b_search_int(dd->onehop_neighbors + dd->onehop_offsets[phelper1], dd->onehop_offsets[phelper1 + 1] - dd->onehop_offsets[phelper1], pvertexid);

                    if (phelper2 > -1) {
                        read[i].exdeg++;
                    }
                }
            }
        }
        else {
            // via removed, update exdeg based on remaining candidates, again lane scheduling should be dynamic
            for (int i = LANE_IDX; i < wd.number_of_members[WIB_IDX]; i += WARP_SIZE) {
                pvertexid = ld.vertices[i].vertexid;

                for (int j = 0; j < wd.removed_count[WIB_IDX]; j++) {
                    phelper1 = dd->removed_candidates[(*dd->wvertices_size * WARP_IDX) + j];
                    phelper2 = d_b_search_int(dd->onehop_neighbors + dd->onehop_offsets[phelper1], dd->onehop_offsets[phelper1 + 1] - dd->onehop_offsets[phelper1], pvertexid);

                    if (phelper2 > -1) {
                        ld.vertices[i].exdeg--;
                    }
                }
            }

            for (int i = LANE_IDX; i < wd.remaining_count[WIB_IDX]; i += WARP_SIZE) {
                pvertexid = read[i].vertexid;

                for (int j = 0; j < wd.removed_count[WIB_IDX]; j++) {
                    phelper1 = dd->removed_candidates[(*dd->wvertices_size * WARP_IDX) + j];
                    phelper2 = d_b_search_int(dd->onehop_neighbors + dd->onehop_offsets[phelper1], dd->onehop_offsets[phelper1 + 1] - dd->onehop_offsets[phelper1], pvertexid);

                    if (phelper2 > -1) {
                        read[i].exdeg--;
                    }
                }
            }
        }
        __syncwarp();

        lane_remaining_count = 0;

        for (int i = LANE_IDX; i < wd.remaining_count[WIB_IDX]; i += WARP_SIZE) {
            if (d_cand_isvalid(read[i], dd, wd, ld)) {
                dd->lane_candidate_indegs[lane_write + lane_remaining_count++] = read[i].indeg;
            }
        }
        __syncwarp();

        // scan to calculate write postion in warp arrays
        phelper2 = lane_remaining_count;
        for (int i = 1; i < WARP_SIZE; i *= 2) {
            phelper1 = __shfl_up_sync(0xFFFFFFFF, lane_remaining_count, i, WARP_SIZE);
            if (LANE_IDX >= i) {
                lane_remaining_count += phelper1;
            }
            __syncwarp();
        }
        // lane remaining count sum is scan for last lane and its value
        if (LANE_IDX == WARP_SIZE - 1) {
            wd.num_val_cands[WIB_IDX] = lane_remaining_count;
        }
        // make scan exclusive
        lane_remaining_count -= phelper2;

        // parallel write lane arrays to warp array
        for (int i = 0; i < phelper2; i++) {
            dd->candidate_indegs[(*dd->wvertices_size * WARP_IDX) + lane_remaining_count + i] = dd->lane_candidate_indegs[lane_write + i];
        }
        __syncwarp();

        d_oe_sort_int(dd->candidate_indegs + (*dd->wvertices_size * WARP_IDX), wd.num_val_cands[WIB_IDX], d_comp_int_desc);

        d_calculate_LU_bounds(dd, wd, ld, wd.num_val_cands[WIB_IDX]);
        if (wd.success[WIB_IDX]) {
            return true;
        }

        // check for failed vertices
        for (int k = LANE_IDX; k < wd.number_of_members[WIB_IDX] && !wd.success[WIB_IDX]; k += WARP_SIZE) {
            if (!d_vert_isextendable(ld.vertices[k], dd, wd, ld)) {
                wd.success[WIB_IDX] = true;
                break;
            }

        }
        __syncwarp();
        if (wd.success[WIB_IDX]) {
            return true;
        }

        lane_remaining_count = 0;
        lane_removed_count = 0;

        // check for failed candidates
        for (int i = LANE_IDX; i < wd.remaining_count[WIB_IDX]; i += WARP_SIZE) {
            if (d_cand_isvalid(read[i], dd, wd, ld)) {
                dd->lane_remaining_candidates[lane_write + lane_remaining_count++] = i;
            }
            else {
                dd->lane_removed_candidates[lane_write + lane_removed_count++] = i;
            }
        }
        __syncwarp();

        // scan to calculate write postion in warp arrays
        phelper2 = lane_remaining_count;
        pvertexid = lane_removed_count;
        for (int i = 1; i < WARP_SIZE; i *= 2) {
            phelper1 = __shfl_up_sync(0xFFFFFFFF, lane_remaining_count, i, WARP_SIZE);
            if (LANE_IDX >= i) {
                lane_remaining_count += phelper1;
            }
            phelper1 = __shfl_up_sync(0xFFFFFFFF, lane_removed_count, i, WARP_SIZE);
            if (LANE_IDX >= i) {
                lane_removed_count += phelper1;
            }
            __syncwarp();
        }
        // lane remaining count sum is scan for last lane and its value
        if (LANE_IDX == WARP_SIZE - 1) {
            wd.num_val_cands[WIB_IDX] = lane_remaining_count;
            wd.removed_count[WIB_IDX] = lane_removed_count;
        }
        // make scan exclusive
        lane_remaining_count -= phelper2;
        lane_removed_count -= pvertexid;

        // parallel write lane arrays to warp array
        for (int i = 0; i < phelper2; i++) {
            write[lane_remaining_count + i] = read[dd->lane_remaining_candidates[lane_write + i]];
        }
        // only need removed if going to be using removed to update degrees
        if (!(wd.num_val_cands[WIB_IDX] < wd.removed_count[WIB_IDX])) {
            for (int i = 0; i < pvertexid; i++) {
                dd->removed_candidates[(*dd->wvertices_size * WARP_IDX) + lane_removed_count + i] = read[dd->lane_removed_candidates[lane_write + i]].vertexid;
            }
        }

        if (LANE_IDX == 0) {
            wd.remaining_count[WIB_IDX] = wd.num_val_cands[WIB_IDX];
            wd.rw_counter[WIB_IDX]++;
        }
    }

    // condense vertices so remaining are after members, only needs to be done if they were not written into vertices last time
    if (wd.rw_counter[WIB_IDX] % 2 == 0) {
        for (int i = LANE_IDX; i < wd.remaining_count[WIB_IDX]; i += WARP_SIZE) {
            ld.vertices[wd.number_of_members[WIB_IDX] + i] = dd->remaining_candidates[(*dd->wvertices_size * WARP_IDX) + i];
        }
    }

    if (LANE_IDX == 0) {
        wd.total_vertices[WIB_IDX] = wd.total_vertices[WIB_IDX] - wd.number_of_candidates[WIB_IDX] + wd.remaining_count[WIB_IDX];
        wd.number_of_candidates[WIB_IDX] = wd.remaining_count[WIB_IDX];
    }

    return false;
}

__device__ void d_calculate_LU_bounds(GPU_Data* dd, Warp_Data& wd, Local_Data& ld, int number_of_candidates)
{
    int index;
    int min_clq_indeg;
    int min_indeg_exdeg;
    int min_clq_totaldeg;
    int sum_clq_indeg;

    // initialize the values of the LU calculation variables to the first vertices values so they can be compared to other vertices without error
    min_clq_indeg = ld.vertices[0].indeg;
    min_indeg_exdeg = ld.vertices[0].exdeg;
    min_clq_totaldeg = ld.vertices[0].indeg + ld.vertices[0].exdeg;
    sum_clq_indeg = 0;

    // each warp also has a copy of these variables to allow for intra-warp comparison of these variables.
    if (LANE_IDX == 0) {
        wd.success[WIB_IDX] = false;

        wd.sum_candidate_indeg[WIB_IDX] = 0;
        wd.tightened_upper_bound[WIB_IDX] = 0;

        wd.min_clq_indeg[WIB_IDX] = ld.vertices[0].indeg;
        wd.min_indeg_exdeg[WIB_IDX] = ld.vertices[0].exdeg;
        wd.min_clq_totaldeg[WIB_IDX] = ld.vertices[0].indeg + ld.vertices[0].exdeg;
        wd.sum_clq_indeg[WIB_IDX] = ld.vertices[0].indeg;

        wd.min_ext_deg[WIB_IDX] = d_get_mindeg(wd.number_of_members[WIB_IDX] + 1, dd);
    }
    __syncwarp();

    // each warp finds these values on their subsection of vertices
    for (index = 1 + LANE_IDX; index < wd.number_of_members[WIB_IDX]; index += WARP_SIZE) {
        sum_clq_indeg += ld.vertices[index].indeg;

        if (ld.vertices[index].indeg < min_clq_indeg) {
            min_clq_indeg = ld.vertices[index].indeg;
            min_indeg_exdeg = ld.vertices[index].exdeg;
        }
        else if (ld.vertices[index].indeg == min_clq_indeg) {
            if (ld.vertices[index].exdeg < min_indeg_exdeg) {
                min_indeg_exdeg = ld.vertices[index].exdeg;
            }
        }

        if (ld.vertices[index].indeg + ld.vertices[index].exdeg < min_clq_totaldeg) {
            min_clq_totaldeg = ld.vertices[index].indeg + ld.vertices[index].exdeg;
        }
    }

    // get sum
    for (int i = 1; i < 32; i *= 2) {
        sum_clq_indeg += __shfl_xor_sync(0xFFFFFFFF, sum_clq_indeg, i);
    }
    if (LANE_IDX == 0) {
        // add to shared memory sum
        wd.sum_clq_indeg[WIB_IDX] += sum_clq_indeg;
    }
    __syncwarp();

    // CRITICAL SECTION - each lane then compares their values to the next to get a warp level value
    for (int i = 0; i < WARP_SIZE; i++) {
        if (LANE_IDX == i) {
            if (min_clq_indeg < wd.min_clq_indeg[WIB_IDX]) {
                wd.min_clq_indeg[WIB_IDX] = min_clq_indeg;
                wd.min_indeg_exdeg[WIB_IDX] = min_indeg_exdeg;
            }
            else if (min_clq_indeg == wd.min_clq_indeg[WIB_IDX]) {
                if (min_indeg_exdeg < wd.min_indeg_exdeg[WIB_IDX]) {
                    wd.min_indeg_exdeg[WIB_IDX] = min_indeg_exdeg;
                }
            }

            if (min_clq_totaldeg < wd.min_clq_totaldeg[WIB_IDX]) {
                wd.min_clq_totaldeg[WIB_IDX] = min_clq_totaldeg;
            }
        }
        __syncwarp();
    }

    // CRITICAL SECTION - only first lane does this as there are little calculations
    if (LANE_IDX == 0) {
        if (wd.min_clq_indeg[WIB_IDX] < dd->minimum_degrees[wd.number_of_members[WIB_IDX]])
        {
            // lower
            wd.lower_bound[WIB_IDX] = d_get_mindeg(wd.number_of_members[WIB_IDX], dd) - min_clq_indeg;

            while (wd.lower_bound[WIB_IDX] <= wd.min_indeg_exdeg[WIB_IDX] && wd.min_clq_indeg[WIB_IDX] + wd.lower_bound[WIB_IDX] <
                dd->minimum_degrees[wd.number_of_members[WIB_IDX] + wd.lower_bound[WIB_IDX]]) {
                wd.lower_bound[WIB_IDX]++;
            }

            if (wd.min_clq_indeg[WIB_IDX] + wd.lower_bound[WIB_IDX] < dd->minimum_degrees[wd.number_of_members[WIB_IDX] + wd.lower_bound[WIB_IDX]]) {
                wd.success[WIB_IDX] = true;
            }

            // upper
            wd.upper_bound[WIB_IDX] = floor(wd.min_clq_totaldeg[WIB_IDX] / (*(dd->minimum_degree_ratio))) + 1 - wd.number_of_members[WIB_IDX];

            if (wd.upper_bound[WIB_IDX] > number_of_candidates) {
                wd.upper_bound[WIB_IDX] = number_of_candidates;
            }

            // tighten
            if (wd.lower_bound[WIB_IDX] < wd.upper_bound[WIB_IDX]) {
                // tighten lower
                for (index = 0; index < wd.lower_bound[WIB_IDX]; index++) {
                    wd.sum_candidate_indeg[WIB_IDX] += dd->candidate_indegs[(*dd->wvertices_size * WARP_IDX) + index];
                }

                while (index < wd.upper_bound[WIB_IDX] && wd.sum_clq_indeg[WIB_IDX] + wd.sum_candidate_indeg[WIB_IDX] < wd.number_of_members[WIB_IDX] *
                    dd->minimum_degrees[wd.number_of_members[WIB_IDX] + index]) {
                    wd.sum_candidate_indeg[WIB_IDX] += dd->candidate_indegs[(*dd->wvertices_size * WARP_IDX) + index];
                    index++;
                }

                if (wd.sum_clq_indeg[WIB_IDX] + wd.sum_candidate_indeg[WIB_IDX] < wd.number_of_members[WIB_IDX] * dd->minimum_degrees[wd.number_of_members[WIB_IDX] + index]) {
                    wd.success[WIB_IDX] = true;
                }
                else {
                    wd.lower_bound[WIB_IDX] = index;

                    wd.tightened_upper_bound[WIB_IDX] = index;

                    while (index < wd.upper_bound[WIB_IDX]) {
                        wd.sum_candidate_indeg[WIB_IDX] += dd->candidate_indegs[(*dd->wvertices_size * WARP_IDX) + index];

                        index++;

                        if (wd.sum_clq_indeg[WIB_IDX] + wd.sum_candidate_indeg[WIB_IDX] >= wd.number_of_members[WIB_IDX] *
                            dd->minimum_degrees[wd.number_of_members[WIB_IDX] + index]) {
                            wd.tightened_upper_bound[WIB_IDX] = index;
                        }
                    }

                    if (wd.upper_bound[WIB_IDX] > wd.tightened_upper_bound[WIB_IDX]) {
                        wd.upper_bound[WIB_IDX] = wd.tightened_upper_bound[WIB_IDX];
                    }

                    if (wd.lower_bound[WIB_IDX] > 1) {
                        wd.min_ext_deg[WIB_IDX] = d_get_mindeg(wd.number_of_members[WIB_IDX] + wd.lower_bound[WIB_IDX], dd);
                    }
                }
            }
        }
        else {
            wd.min_ext_deg[WIB_IDX] = d_get_mindeg(wd.number_of_members[WIB_IDX] + 1,
                dd);

            wd.upper_bound[WIB_IDX] = number_of_candidates;

            if (wd.number_of_members[WIB_IDX] < (*(dd->minimum_clique_size))) {
                wd.lower_bound[WIB_IDX] = (*(dd->minimum_clique_size)) - wd.number_of_members[WIB_IDX];
            }
            else {
                wd.lower_bound[WIB_IDX] = 0;
            }
        }

        if (wd.number_of_members[WIB_IDX] + wd.upper_bound[WIB_IDX] < (*(dd->minimum_clique_size))) {
            wd.success[WIB_IDX] = true;
        }

        if (wd.upper_bound[WIB_IDX] < 0 || wd.upper_bound[WIB_IDX] < wd.lower_bound[WIB_IDX]) {
            wd.success[WIB_IDX] = true;
        }
    }
    __syncwarp();
}

__device__ void d_check_for_clique(GPU_Data* dd, Warp_Data& wd, Local_Data& ld)
{
    bool clique;

    clique = true;

    for (int k = LANE_IDX; k < wd.number_of_members[WIB_IDX]; k += WARP_SIZE) {
        if (ld.vertices[k].indeg < dd->minimum_degrees[wd.number_of_members[WIB_IDX]]) {
            clique = false;
            break;
        }
    }
    // set to false if any threads in warp do not meet degree requirement
    clique = !(__any_sync(0xFFFFFFFF, !clique));

    // if clique write to warp buffer for cliques
    if (clique) {
        uint64_t start_write = (*dd->wcliques_size * WARP_IDX) + dd->wcliques_offset[(*dd->wcliques_offset_size * WARP_IDX) + dd->wcliques_count[WARP_IDX]];
        for (int k = LANE_IDX; k < wd.number_of_members[WIB_IDX]; k += WARP_SIZE) {
            dd->wcliques_vertex[start_write + k] = ld.vertices[k].vertexid;
        }
        if (LANE_IDX == 0) {
            (dd->wcliques_count[WARP_IDX])++;
            dd->wcliques_offset[*dd->wcliques_offset_size * WARP_IDX + dd->wcliques_count[WARP_IDX]] = start_write - (*dd->wcliques_size * WARP_IDX) + wd.number_of_members[WIB_IDX];
        }
    }
}

__device__ void d_write_to_tasks(GPU_Data* dd, Warp_Data& wd, Local_Data& ld)
{
    uint64_t start_write;

    start_write = (*dd->wtasks_size * WARP_IDX) + dd->wtasks_offset[*dd->wtasks_offset_size * WARP_IDX + dd->wtasks_count[WARP_IDX]];

    for (int k = LANE_IDX; k < wd.total_vertices[WIB_IDX]; k += WARP_SIZE) {
        dd->wtasks_vertices[start_write + k].vertexid = ld.vertices[k].vertexid;
        dd->wtasks_vertices[start_write + k].label = ld.vertices[k].label;
        dd->wtasks_vertices[start_write + k].indeg = ld.vertices[k].indeg;
        dd->wtasks_vertices[start_write + k].exdeg = ld.vertices[k].exdeg;
        dd->wtasks_vertices[start_write + k].lvl2adj = 0;
    }
    if (LANE_IDX == 0) {
        dd->wtasks_count[WARP_IDX]++;
        dd->wtasks_offset[(*dd->wtasks_offset_size * WARP_IDX) + dd->wtasks_count[WARP_IDX]] = start_write - (*dd->wtasks_size * WARP_IDX) + wd.total_vertices[WIB_IDX];
    }
}

// --- TERTIARY KENERLS ---
// searches an int array for a certain int, returns the position in the array that item was found, or -1 if not found
__device__ int d_b_search_int(int* search_array, int array_size, int search_number)
{
    // ALGO - BINARY
    // TYPE - SERIAL
    // SPEED - O(log(n))
    
    int low;
    int high;
    int mid;
    int mid_value;
    int comp;

    low = 0;
    high = array_size - 1;

    while (low < high) {
        mid = (low + high) / 2;
        mid_value = search_array[mid];
        comp = (mid_value < search_number);

        low = low + comp * (mid + 1 - low);
        high = high - !comp * (high - mid);
    }

    // Now low == high, check if it's the search_number
    return (search_array[low] == search_number) ? low : -1;
}

// consider using merge
__device__ void d_oe_sort_vert(Vertex* target, int size, int (*func)(Vertex&, Vertex&))
{
    // ALGO - ODD/EVEN
    // TYPE - PARALLEL
    // SPEED - O(n^2)

    Vertex vertex1;
    Vertex vertex2;

    for (int i = 0; i < size; i++) {
        for (int j = (i % 2) + (LANE_IDX * 2); j < size - 1; j += (WARP_SIZE * 2)) {
            vertex1 = target[j];
            vertex2 = target[j + 1];

            if (func(vertex1, vertex2) == 1) {
                target[j] = vertex2;
                target[j + 1] = vertex1;
            }
        }
        __syncwarp();
    }
}

__device__ void d_oe_sort_int(int* target, int size, int (*func)(int, int))
{
    // ALGO - ODD/EVEN
    // TYPE - PARALLEL
    // SPEED - O(n^2)

    int num1;
    int num2;

    for (int i = 0; i < size; i++) {
        for (int j = (i % 2) + (LANE_IDX * 2); j < size - 1; j += (WARP_SIZE * 2)) {
            num1 = target[j];
            num2 = target[j + 1];

            if (func(num1, num2) == 1) {
                target[j] = num2;
                target[j + 1] = num1;
            }
        }
        __syncwarp();
    }
}

// --- DEBUG KERNELS ---
__device__ void d_print_vertices(Vertex* vertices, int size)
{
    printf("\nOffsets:\n0 %i\nVertex:\n", size);
    for (int i = 0; i < size; i++) {
        printf("%i ", vertices[i].vertexid);
    }
    printf("\nLabel:\n");
    for (int i = 0; i < size; i++) {
        printf("%i ", vertices[i].label);
    }
    printf("\nIndeg:\n");
    for (int i = 0; i < size; i++) {
        printf("%i ", vertices[i].indeg);
    }
    printf("\nExdeg:\n");
    for (int i = 0; i < size; i++) {
        printf("%i ", vertices[i].exdeg);
    }
    printf("\nLvl2adj:\n");
    for (int i = 0; i < size; i++) {
        printf("%i ", vertices[i].lvl2adj);
    }
    printf("\n");
}