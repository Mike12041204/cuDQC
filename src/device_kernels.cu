#include "hip/hip_runtime.h"
#include "../inc/common.hpp"
#include "../inc/device_kernels.hpp"

// --- PRIMARY KERNELS ---
__global__ void d_expand_level(GPU_Data* dd)
{
    __shared__ Warp_Data wd;        // data is stored in data structures to reduce the number of variables that need to be passed to methods
    Local_Data ld;
    int num_mem;                    // helper variables, not passed through to any methods
    int index;

    // --- CURRENT LEVEL ---

    // reset warp tasks and cliques counts
    if (LANE_IDX == 0) {
        dd->wtasks_count[WARP_IDX] = 0;
        dd->wcliques_count[WARP_IDX] = 0;
    }
    __syncwarp();

    // initialize i for each warp
    int i = WARP_IDX;
    
    while (i < *dd->tasks_count) {

        // INITIALIZE OLD VERTICES
        // get information on vertices being handled within tasks
        if (LANE_IDX == 0) {
            wd.start[WIB_IDX] = dd->tasks_offset[i];
            wd.end[WIB_IDX] = dd->tasks_offset[i + 1];
            wd.tot_vert[WIB_IDX] = wd.end[WIB_IDX] - wd.start[WIB_IDX];
        }
        __syncwarp();

        // each warp gets partial number of members
        num_mem = 0;
        for (uint64_t j = wd.start[WIB_IDX] + LANE_IDX; j < wd.end[WIB_IDX]; j += WARP_SIZE) {
            if (dd->tasks_vertices[j].label != 1) {
                break;
            }
            num_mem++;
        }
        // sum members across warp
        for (int k = 1; k < 32; k *= 2) {
            num_mem += __shfl_xor_sync(0xFFFFFFFF, num_mem, k);
        }

        if (LANE_IDX == 0) {
            wd.num_mem[WIB_IDX] = num_mem;
            wd.num_cand[WIB_IDX] = wd.tot_vert[WIB_IDX] - wd.num_mem[WIB_IDX];
            wd.expansions[WIB_IDX] = wd.num_cand[WIB_IDX];
        }
        __syncwarp();

        // LOOKAHEAD PRUNING
        if(LANE_IDX == 0){
            wd.success[WIB_IDX] = true;
        }
        __syncwarp();

        // sets success to false if lookahead works
        d_lookahead_pruning(dd, wd, ld);
        
        if (wd.success[WIB_IDX]) {
            // schedule warps next task
            if (LANE_IDX == 0) {
                i = atomicAdd(dd->current_task, 1);
            }
            i = __shfl_sync(0xFFFFFFFF, i, 0);
            continue;
        }

        // --- NEXT LEVEL ---
        for (int j = 0; j < wd.expansions[WIB_IDX]; j++)
        {

            // REMOVE ONE VERTEX
            if (j > 0) {
                if(LANE_IDX == 0){
                    wd.success[WIB_IDX] = true;
                }
                __syncwarp();

                // set success to false is failed vertex found 
                d_remove_one_vertex(dd, wd, ld);

                if (!wd.success[WIB_IDX]) {
                    break;
                }
            }

            // INITIALIZE NEW VERTICES
            if (LANE_IDX == 0) {
                wd.number_of_members[WIB_IDX] = wd.num_mem[WIB_IDX];
                wd.number_of_candidates[WIB_IDX] = wd.num_cand[WIB_IDX];
                wd.total_vertices[WIB_IDX] = wd.tot_vert[WIB_IDX];
            }
            __syncwarp();

            // select whether to store vertices in global or shared memory based on size
            if (wd.total_vertices[WIB_IDX] <= VERTICES_SIZE) {
                ld.vertices = wd.shared_vertices + (VERTICES_SIZE * WIB_IDX);
            }
            else {
                ld.vertices = dd->global_vertices + (*dd->WVERTICES_SIZE * WARP_IDX);
            }

            // copy vertices
            for (index = LANE_IDX; index < wd.number_of_members[WIB_IDX]; index += WARP_SIZE) {
                ld.vertices[index] = dd->tasks_vertices[wd.start[WIB_IDX] + index];
            }
            for (; index < wd.total_vertices[WIB_IDX] - 1; index += WARP_SIZE) {
                ld.vertices[index + 1] = dd->tasks_vertices[wd.start[WIB_IDX] + index];
            }
            if (LANE_IDX == 0) {
                ld.vertices[wd.number_of_members[WIB_IDX]] = dd->tasks_vertices[wd.start[WIB_IDX] + 
                    wd.total_vertices[WIB_IDX] - 1];
            }
            __syncwarp();

            // ADD ONE VERTEX
            if(LANE_IDX == 0){
                wd.success[WIB_IDX] = true;
            }
            __syncwarp();
            
            // sets success to false if failed found
            d_add_one_vertex(dd, wd, ld);

            // if failed found check for clique and continue on to the next iteration
            if (!wd.success[WIB_IDX]) {
                d_check_for_clique(dd, wd, ld);
                continue;
            }

            // CRITICAL VERTEX PRUNING
            if(LANE_IDX == 0){
                wd.success[WIB_IDX] = 0;
            }
            __syncwarp();

            // sets success to 2 if critical failure, 1 if failed found
            d_critical_vertex_pruning(dd, wd, ld);

            // critical fail, cannot be clique continue onto next iteration
            if (wd.success[WIB_IDX] == 2) {
                continue;
            }

            // HANDLE CLIQUES
            d_check_for_clique(dd, wd, ld);

            // if vertex in x found as not extendable continue to next iteration
            if (wd.success[WIB_IDX] == 1) {
                continue;
            }

            // WRITE TASKS TO BUFFERS
            // sort vertices in Quick efficient enumeration order before writing
            d_oe_sort_vert(ld.vertices, wd.total_vertices[WIB_IDX], d_comp_vert_Q);

            if (wd.number_of_candidates[WIB_IDX] > 0) {
                d_write_to_tasks(dd, wd, ld);
            }
        }

        // schedule warps next task
        if (LANE_IDX == 0) {
            i = atomicAdd(dd->current_task, 1);
        }
        i = __shfl_sync(0xFFFFFFFF, i, 0);
    }

    if (LANE_IDX == 0) {
        // sum to find tasks count
        atomicAdd(dd->total_tasks, dd->wtasks_count[WARP_IDX]);
        atomicAdd(dd->total_cliques, dd->wcliques_count[WARP_IDX]);
    }

    // TODO - this should be easy to remove and just make local in transfer_buffers
    if (IDX == 0) {
        *dd->buffer_offset_start = *dd->buffer_count + 1;
        *dd->buffer_start = dd->buffer_offset[*dd->buffer_count];
        *dd->cliques_offset_start = *dd->cliques_count + 1;
        *dd->cliques_start = dd->cliques_offset[*dd->cliques_count];
    }
}

__global__ void d_transfer_buffers(GPU_Data* dd, uint64_t* tasks_count, uint64_t* buffer_count, 
                                   uint64_t* cliques_count)
{
    __shared__ uint64_t tasks_write[WARPS_PER_BLOCK];
    __shared__ int tasks_offset_write[WARPS_PER_BLOCK];
    __shared__ uint64_t cliques_write[WARPS_PER_BLOCK];
    __shared__ int cliques_offset_write[WARPS_PER_BLOCK];
    __shared__ int twarp;
    __shared__ int toffsetwrite;
    __shared__ int twrite;
    __shared__ int tasks_end;

    // point of this is to find how many vertices will be transfered to tasks, it is easy to know how many tasks as it will just
    // be the expansion threshold, but to find how many vertices we must now the total size of all the tasks that will be copied.
    // each block does this but really could be done by one thread outside the GPU
    if (TIB_IDX == 0) {
        toffsetwrite = 0;
        twrite = 0;

        for (int i = 0; i < NUMBER_OF_WARPS; i++) {
            // if next warps count is more than expand threshold mark as such and break
            if (toffsetwrite + dd->wtasks_count[i] >= *dd->EXPAND_THRESHOLD) {
                twarp = i;
                break;
            }
            // else adds its size and count
            twrite += dd->wtasks_offset[(*dd->WTASKS_OFFSET_SIZE * i) + dd->wtasks_count[i]];
            toffsetwrite += dd->wtasks_count[i];
        }
        // final size is the size of all tasks up until last warp and the remaining tasks in the last warp until expand threshold is satisfied
        tasks_end = twrite + dd->wtasks_offset[(*dd->WTASKS_OFFSET_SIZE * twarp) + (*dd->EXPAND_THRESHOLD - toffsetwrite)];
    }
    __syncthreads();

    // warp level
    if (LANE_IDX == 0) {
        tasks_write[WIB_IDX] = 0;
        tasks_offset_write[WIB_IDX] = 1;
        cliques_write[WIB_IDX] = 0;
        cliques_offset_write[WIB_IDX] = 1;

        for (int i = 0; i < WARP_IDX; i++) {
            tasks_offset_write[WIB_IDX] += dd->wtasks_count[i];
            tasks_write[WIB_IDX] += dd->wtasks_offset[(*dd->WTASKS_OFFSET_SIZE * i) + dd->wtasks_count[i]];

            cliques_offset_write[WIB_IDX] += dd->wcliques_count[i];
            cliques_write[WIB_IDX] += dd->wcliques_offset[(*dd->WCLIQUES_OFFSET_SIZE * i) + dd->wcliques_count[i]];
        }
    }
    __syncwarp();
    
    // move to tasks and buffer
    for (int i = LANE_IDX + 1; i <= dd->wtasks_count[WARP_IDX]; i += WARP_SIZE) {
        if (tasks_offset_write[WIB_IDX] + i - 1 <= *dd->EXPAND_THRESHOLD) {
            // to tasks
            dd->tasks_offset[tasks_offset_write[WIB_IDX] + i - 1] = dd->wtasks_offset[(*dd->WTASKS_OFFSET_SIZE * WARP_IDX) + i] + tasks_write[WIB_IDX];
        }
        else {
            // to buffer
            dd->buffer_offset[tasks_offset_write[WIB_IDX] + i - 2 - *dd->EXPAND_THRESHOLD + *dd->buffer_offset_start] = dd->wtasks_offset[(*dd->WTASKS_OFFSET_SIZE * WARP_IDX) + i] +
                tasks_write[WIB_IDX] - tasks_end + *dd->buffer_start;
        }
    }

    for (int i = LANE_IDX; i < dd->wtasks_offset[(*dd->WTASKS_OFFSET_SIZE * WARP_IDX) + dd->wtasks_count[WARP_IDX]]; i += WARP_SIZE) {
        if (tasks_write[WIB_IDX] + i < tasks_end) {
            // to tasks
            dd->tasks_vertices[tasks_write[WIB_IDX] + i] = dd->wtasks_vertices[(*dd->WTASKS_SIZE * WARP_IDX) + i];
        }
        else {
            // to buffer
            dd->buffer_vertices[*dd->buffer_start + tasks_write[WIB_IDX] + i - tasks_end] = dd->wtasks_vertices[(*dd->WTASKS_SIZE * WARP_IDX) + i];
        }
    }
    // NOTE - this sync is important for some reason, larger graphs/et dont work without it
    __syncthreads();

    //move to cliques
    for (int i = LANE_IDX + 1; i <= dd->wcliques_count[WARP_IDX]; i += WARP_SIZE) {
        dd->cliques_offset[*dd->cliques_offset_start + cliques_offset_write[WIB_IDX] + i - 2] = dd->wcliques_offset[(*dd->WCLIQUES_OFFSET_SIZE * WARP_IDX) + i] + *dd->cliques_start + 
            cliques_write[WIB_IDX];
    }
    for (int i = LANE_IDX; i < dd->wcliques_offset[(*dd->WCLIQUES_OFFSET_SIZE * WARP_IDX) + dd->wcliques_count[WARP_IDX]]; i += WARP_SIZE) {
        dd->cliques_vertex[*dd->cliques_start + cliques_write[WIB_IDX] + i] = dd->wcliques_vertex[(*dd->WCLIQUES_SIZE * WARP_IDX) + i];
    }

    if (IDX == 0) {
        // handle tasks and buffer counts
        if (*dd->total_tasks <= *dd->EXPAND_THRESHOLD) {
            *dd->tasks_count = *dd->total_tasks;
        }
        else {
            *dd->tasks_count = *dd->EXPAND_THRESHOLD;
            *dd->buffer_count += *dd->total_tasks - *dd->EXPAND_THRESHOLD;
        }
        *dd->cliques_count += *dd->total_cliques;

        *dd->total_tasks = 0;
        *dd->total_cliques = 0;
        (*dd->current_level)++;

        *dd->current_task = NUMBER_OF_WARPS;
        *tasks_count = *dd->tasks_count;
        *buffer_count = *dd->buffer_count;
        *cliques_count = *dd->cliques_count;
    }
}

__global__ void d_fill_from_buffer(GPU_Data* dd, uint64_t* buffer_count)
{
    // get read and write locations
    int write_amount = (*dd->buffer_count >= *dd->EXPAND_THRESHOLD - *dd->tasks_count) ? *dd->EXPAND_THRESHOLD - *dd->tasks_count : *dd->buffer_count;
    uint64_t start_buffer = dd->buffer_offset[*dd->buffer_count - write_amount];
    uint64_t end_buffer = dd->buffer_offset[*dd->buffer_count];
    uint64_t size_buffer = end_buffer - start_buffer;
    uint64_t start_write = dd->tasks_offset[*dd->tasks_count];

    // handle offsets
    for (int i = IDX + 1; i <= write_amount; i += NUMBER_OF_DTHREADS) {
        dd->tasks_offset[*dd->tasks_count + i] = start_write + dd->buffer_offset[*dd->buffer_count - write_amount + i] - start_buffer;
    }

    // handle data
    for (int i = IDX; i < size_buffer; i += NUMBER_OF_DTHREADS) {
        dd->tasks_vertices[start_write + i] = dd->buffer_vertices[start_buffer + i];
    }

    if (IDX == 0) {
        *dd->tasks_count += write_amount;
        *dd->buffer_count -= write_amount;

        *buffer_count = *dd->buffer_count;
    }
}

// --- SECONDARY EXPANSION KERNELS ---
// DQC - implement, also set success to false is lookahead works else true
// TODO - make a write clique method
__device__ void d_lookahead_pruning(GPU_Data* dd, Warp_Data& wd, Local_Data& ld)
{
    // int pvertexid;
    // int phelper1;
    // int phelper2;
    // uint64_t start_write;

    // DQC - when method is implemented to return properly this can be removed
    if (LANE_IDX == 0) {
        wd.success[WIB_IDX] = false;
    }
    __syncwarp();

    // // check if members meet degree requirement, dont need to check 2hop adj as diameter pruning guarentees all members will be within 2hops of eveything
    // for (int i = LANE_IDX; i < wd.num_mem[WIB_IDX] && wd.success[WIB_IDX]; i += WARP_SIZE) {
    //     if (dd->tasks_vertices[wd.start[WIB_IDX] + i].indeg + dd->tasks_vertices[wd.start[WIB_IDX] + i].exdeg < dd->minimum_degrees[wd.tot_vert[WIB_IDX]]) {
    //         wd.success[WIB_IDX] = false;
    //         break;
    //     }
    // }
    // __syncwarp();

    // if (!wd.success[WIB_IDX]) {
    //     return 0;
    // }

    // // update lvl2adj to candidates for all vertices
    // for (int i = wd.num_mem[WIB_IDX] + LANE_IDX; i < wd.tot_vert[WIB_IDX]; i += WARP_SIZE) {
    //     pvertexid = dd->tasks_vertices[wd.start[WIB_IDX] + i].vertexid;
        
    //     for (int j = wd.num_mem[WIB_IDX]; j < wd.tot_vert[WIB_IDX]; j++) {
    //         if (j == i) {
    //             continue;
    //         }

    //         phelper1 = dd->tasks_vertices[wd.start[WIB_IDX] + j].vertexid;
    //         phelper2 = d_b_search_int(dd->twohop_neighbors + dd->twohop_offsets[phelper1], dd->twohop_offsets[phelper1 + 1] - dd->twohop_offsets[phelper1], pvertexid);
        
    //         if (phelper2 > -1) {
    //             dd->tasks_vertices[wd.start[WIB_IDX] + i].lvl2adj++;
    //         }
    //     }
    // }
    // __syncwarp();

    // // compares all vertices to the lemmas from Quick
    // for (int j = wd.num_mem[WIB_IDX] + LANE_IDX; j < wd.tot_vert[WIB_IDX] && wd.success[WIB_IDX]; j += WARP_SIZE) {
    //     if (dd->tasks_vertices[wd.start[WIB_IDX] + j].lvl2adj < wd.num_cand[WIB_IDX] - 1 || dd->tasks_vertices[wd.start[WIB_IDX] + j].indeg + dd->tasks_vertices[wd.start[WIB_IDX] + j].exdeg < dd->minimum_degrees[wd.tot_vert[WIB_IDX]]) {
    //         wd.success[WIB_IDX] = false;
    //         break;
    //     }
    // }
    // __syncwarp();

    // if (wd.success[WIB_IDX]) {
    //     // write to cliques
    //     start_write = (*dd->WCLIQUES_SIZE * WARP_IDX) + dd->wcliques_offset[(*dd->WCLIQUES_OFFSET_SIZE * WARP_IDX) + dd->wcliques_count[WARP_IDX]];
    //     for (int j = LANE_IDX; j < wd.tot_vert[WIB_IDX]; j += WARP_SIZE) {
    //         dd->wcliques_vertex[start_write + j] = dd->tasks_vertices[wd.start[WIB_IDX] + j].vertexid;
    //     }
    //     if (LANE_IDX == 0) {
    //         (dd->wcliques_count[WARP_IDX])++;
    //         dd->wcliques_offset[(*dd->WCLIQUES_OFFSET_SIZE * WARP_IDX) + dd->wcliques_count[WARP_IDX]] = start_write - (*dd->WCLIQUES_SIZE * WARP_IDX) + wd.tot_vert[WIB_IDX];
    //     }
    //     return 1;
    // }

    // return 0;
}

// sets success to false if failed found else it remains true
__device__ void d_remove_one_vertex(GPU_Data* dd, Warp_Data& wd, Local_Data& ld)
{
    int pvertexid;
    int phelper1;
    int phelper2;
    uint64_t pneighbors_start;
    uint64_t pneighbors_end;
    uint64_t pneighbors_size;
    int min_out_deg;
    int min_in_deg;

    min_out_deg = d_get_mindeg(wd.num_mem[WIB_IDX], dd->minimum_out_degrees, 
                               *dd->minimum_clique_size);
    min_in_deg = d_get_mindeg(wd.num_mem[WIB_IDX], dd->minimum_in_degrees, 
                               *dd->minimum_clique_size);

    // remove the last candidate in vertices
    if (LANE_IDX == 0) {
        wd.num_cand[WIB_IDX]--;
        wd.tot_vert[WIB_IDX]--;
    }
    __syncwarp();

    // update info of vertices connected to removed cand
    pvertexid = dd->tasks_vertices[wd.start[WIB_IDX] + wd.tot_vert[WIB_IDX]].vertexid;

    pneighbors_start = dd->out_offsets[pvertexid];
    pneighbors_end = dd->out_offsets[pvertexid + 1];
    pneighbors_size = pneighbors_end - pneighbors_start;

    for (int i = LANE_IDX; i < wd.tot_vert[WIB_IDX] && wd.success[WIB_IDX]; i += WARP_SIZE) {
        
        phelper1 = dd->tasks_vertices[wd.start[WIB_IDX] + i].vertexid;
        phelper2 = d_b_search_int(dd->out_neighbors + pneighbors_start, pneighbors_size, phelper1);

        if (phelper2 > -1) {
            dd->tasks_vertices[wd.start[WIB_IDX] + i].in_can_deg--;

            if (phelper2 < wd.num_mem[WIB_IDX] && 
                dd->tasks_vertices[wd.start[WIB_IDX] + i].in_mem_deg + 
                dd->tasks_vertices[wd.start[WIB_IDX] + i].in_can_deg < min_in_deg) {
                
                wd.success[WIB_IDX] = false;
                break;
            }
        }
    }
    __syncwarp();

    if (!wd.success[WIB_IDX]) {
        return;
    }

    pneighbors_start = dd->in_offsets[pvertexid];
    pneighbors_end = dd->in_offsets[pvertexid + 1];
    pneighbors_size = pneighbors_end - pneighbors_start;

    for (int i = LANE_IDX; i < wd.tot_vert[WIB_IDX] && wd.success[WIB_IDX]; i += WARP_SIZE) {
        
        phelper1 = dd->tasks_vertices[wd.start[WIB_IDX] + i].vertexid;
        phelper2 = d_b_search_int(dd->in_neighbors + pneighbors_start, pneighbors_size, phelper1);

        if (phelper2 > -1) {
            dd->tasks_vertices[wd.start[WIB_IDX] + i].out_can_deg--;

            if (phelper2 < wd.num_mem[WIB_IDX] && 
                dd->tasks_vertices[wd.start[WIB_IDX] + i].out_mem_deg + 
                dd->tasks_vertices[wd.start[WIB_IDX] + i].out_can_deg < min_out_deg) {
                
                wd.success[WIB_IDX] = false;
                break;
            }
        }
    }
    __syncwarp();
}

// sets success to false if failed found else leaves as true
__device__ void d_add_one_vertex(GPU_Data* dd, Warp_Data& wd, Local_Data& ld)
{
    int pvertexid;
    int phelper1;
    int phelper2;
    uint64_t pneighbors_start;
    uint64_t pneighbors_end;
    uint64_t pneighbors_size;
    int min_out_deg;
    int min_in_deg;

    min_out_deg = d_get_mindeg(wd.number_of_members[WIB_IDX] + 2, dd->minimum_out_degrees, 
                               *dd->minimum_clique_size);
    min_in_deg = d_get_mindeg(wd.number_of_members[WIB_IDX] + 2, dd->minimum_in_degrees, 
                               *dd->minimum_clique_size);

    // ADD ONE VERTEX
    pvertexid = ld.vertices[wd.number_of_members[WIB_IDX]].vertexid;

    if (LANE_IDX == 0) {
        ld.vertices[wd.number_of_members[WIB_IDX]].label = 1;
        wd.number_of_members[WIB_IDX]++;
        wd.number_of_candidates[WIB_IDX]--;
    }
    __syncwarp();

    // update degrees of adjacent vertices
    pneighbors_start = dd->out_offsets[pvertexid];
    pneighbors_end = dd->out_offsets[pvertexid + 1];
    pneighbors_size = pneighbors_end - pneighbors_start;

    for (int i = LANE_IDX; i < wd.total_vertices[WIB_IDX]; i += WARP_SIZE) {
        
        phelper1 = ld.vertices[i].vertexid;
        phelper2 = d_b_search_int(dd->out_neighbors + pneighbors_start, pneighbors_size, 
                                  phelper1);

        if (phelper2 > -1) {
            ld.vertices[i].in_mem_deg++;
            ld.vertices[i].in_can_deg--;
        }
    }

    pneighbors_start = dd->in_offsets[pvertexid];
    pneighbors_end = dd->in_offsets[pvertexid + 1];
    pneighbors_size = pneighbors_end - pneighbors_start;

    for (int i = LANE_IDX; i < wd.total_vertices[WIB_IDX]; i += WARP_SIZE) {
        
        phelper1 = ld.vertices[i].vertexid;
        phelper2 = d_b_search_int(dd->in_neighbors + pneighbors_start, pneighbors_size, 
                                  phelper1);

        if (phelper2 > -1) {
            ld.vertices[i].out_mem_deg++;
            ld.vertices[i].out_can_deg--;
        }
    }
    __syncwarp();

    // DIAMETER PRUNING
    d_diameter_pruning(dd, wd, ld, pvertexid, min_out_deg, min_in_deg);

    // DEGREE BASED PRUNING
    // sets success to false if failed found else leaves as true
    d_degree_pruning(dd, wd, ld);
}

// sets success as 2 if critical fail, 1 if failed found or invalid bound, 0 otherwise
// DQC - implement
__device__ int d_critical_vertex_pruning(GPU_Data* dd, Warp_Data& wd, Local_Data& ld)
{
    // int phelper1;                   // intersection
    // int number_of_crit_adj;         // pruning
    // bool failed_found;

    // // CRITICAL VERTEX PRUNING 
    // // iterate through all vertices in clique
    // for (int k = 0; k < wd.number_of_members[WIB_IDX]; k++) {

    //     // if they are a critical vertex
    //     if (ld.vertices[k].indeg + ld.vertices[k].exdeg == dd->minimum_degrees[wd.number_of_members[WIB_IDX] + wd.lower_bound[WIB_IDX]] && ld.vertices[k].exdeg > 0) {
    //         phelper1 = ld.vertices[k].vertexid;

    //         // iterate through all candidates
    //         for (int i = wd.number_of_members[WIB_IDX] + LANE_IDX; i < wd.total_vertices[WIB_IDX]; i += WARP_SIZE) {
    //             if (ld.vertices[i].label != 4) {
    //                 // if candidate is neighbor of critical vertex mark as such
    //                 if (d_b_search_int(dd->onehop_neighbors + dd->onehop_offsets[phelper1], dd->onehop_offsets[phelper1 + 1] - dd->onehop_offsets[phelper1], ld.vertices[i].vertexid) > -1) {
    //                     ld.vertices[i].label = 4;
    //                 }
    //             }
    //         }
    //     }
    //     __syncwarp();
    // }

    // // sort vertices so that critical vertex adjacent candidates are immediately after vertices within the clique
    // d_oe_sort_vert(ld.vertices + wd.number_of_members[WIB_IDX], wd.number_of_candidates[WIB_IDX], d_comp_vert_cv);

    // // count number of critical adjacent vertices
    // number_of_crit_adj = 0;
    // for (int i = wd.number_of_members[WIB_IDX] + LANE_IDX; i < wd.total_vertices[WIB_IDX]; i += WARP_SIZE) {
    //     if (ld.vertices[i].label == 4) {
    //         number_of_crit_adj++;
    //     }
    //     else {
    //         break;
    //     }
    // }
    // // get sum
    // for (int i = 1; i < 32; i *= 2) {
    //     number_of_crit_adj += __shfl_xor_sync(0xFFFFFFFF, number_of_crit_adj, i);
    // }

    // failed_found = false;

    // // reset adjacencies
    // for (int i = LANE_IDX; i < wd.total_vertices[WIB_IDX]; i += WARP_SIZE) {
    //     dd->adjacencies[(*dd->WVERTICES_SIZE * WARP_IDX) + i] = 0;
    // }

    // // if there were any neighbors of critical vertices
    // if (number_of_crit_adj > 0)
    // {
    //     // iterate through all vertices and update their degrees as if critical adjacencies were added and keep track of how many critical adjacencies they are adjacent to
    //     for (int k = LANE_IDX; k < wd.total_vertices[WIB_IDX]; k += WARP_SIZE) {
    //         phelper1 = ld.vertices[k].vertexid;

    //         for (int i = wd.number_of_members[WIB_IDX]; i < wd.number_of_members[WIB_IDX] + number_of_crit_adj; i++) {
    //             if (d_b_search_int(dd->onehop_neighbors + dd->onehop_offsets[phelper1], dd->onehop_offsets[phelper1 + 1] - dd->onehop_offsets[phelper1], ld.vertices[i].vertexid) > -1) {
    //                 ld.vertices[k].indeg++;
    //                 ld.vertices[k].exdeg--;
    //             }

    //             if (d_b_search_int(dd->twohop_neighbors + dd->twohop_offsets[phelper1], dd->twohop_offsets[phelper1 + 1] - dd->twohop_offsets[phelper1], ld.vertices[i].vertexid) > -1) {
    //                 dd->adjacencies[(*dd->WVERTICES_SIZE * WARP_IDX) + k]++;
    //             }
    //         }
    //     }
    //     __syncwarp();

    //     // all vertices within the clique must be within 2hops of the newly added critical vertex adj vertices
    //     for (int k = LANE_IDX; k < wd.number_of_members[WIB_IDX]; k += WARP_SIZE) {
    //         if (dd->adjacencies[(*dd->WVERTICES_SIZE * WARP_IDX) + k] != number_of_crit_adj) {
    //             failed_found = true;
    //             break;
    //         }
    //     }
    //     failed_found = __any_sync(0xFFFFFFFF, failed_found);
    //     if (failed_found) {
    //         return 2;
    //     }

    //     // all critical adj vertices must all be within 2 hops of each other
    //     for (int k = wd.number_of_members[WIB_IDX] + LANE_IDX; k < wd.number_of_members[WIB_IDX] + number_of_crit_adj; k += WARP_SIZE) {
    //         if (dd->adjacencies[(*dd->WVERTICES_SIZE * WARP_IDX) + k] < number_of_crit_adj - 1) {
    //             failed_found = true;
    //             break;
    //         }
    //     }
    //     failed_found = __any_sync(0xFFFFFFFF, failed_found);
    //     if (failed_found) {
    //         return 2;
    //     }

    //     // no failed vertices found so add all critical vertex adj candidates to clique
    //     for (int k = wd.number_of_members[WIB_IDX] + LANE_IDX; k < wd.number_of_members[WIB_IDX] + number_of_crit_adj; k += WARP_SIZE) {
    //         ld.vertices[k].label = 1;
    //     }

    //     if (LANE_IDX == 0) {
    //         wd.number_of_members[WIB_IDX] += number_of_crit_adj;
    //         wd.number_of_candidates[WIB_IDX] -= number_of_crit_adj;
    //     }
    //     __syncwarp();
    // }

    // // DIAMTER PRUNING
    // d_diameter_pruning_cv(dd, wd, ld, number_of_crit_adj);

    // // DEGREE BASED PRUNING
    // failed_found = d_degree_pruning(dd, wd, ld);

    // // if vertex in x found as not extendable continue to next iteration
    // if (failed_found) {
    //     return 1;
    // }

    // return 0;
}

// diameter pruning intitializes vertices labels and candidate indegs array for use in iterative 
// degree pruning
__device__ void d_diameter_pruning(GPU_Data* dd, Warp_Data& wd, Local_Data& ld, int pvertexid, 
                                   int min_out_deg, int min_in_deg)
{
    int lane_write;
    int phelper1;                       // intersection
    int phelper2;
    int lane_remaining_count;           // vertex iteration
    uint64_t pneighbors_start;
    uint64_t pneighbors_end;
    uint64_t pneighbors_size;

    lane_write = (*dd->WVERTICES_SIZE * WARP_IDX) + ((*dd->WVERTICES_SIZE / WARP_SIZE) * LANE_IDX);
    lane_remaining_count = 0;

    // set all candidates as invalid
    for (int i = wd.number_of_members[WIB_IDX] + LANE_IDX; i < wd.total_vertices[WIB_IDX]; 
         i += WARP_SIZE) {
        
        ld.vertices[i].label = -1;
    }
    __syncwarp();

    // mark all candidates within two hops of added vertex as valid
    pneighbors_start = dd->twohop_offsets[pvertexid];
    pneighbors_end = dd->twohop_offsets[pvertexid + 1];
    pneighbors_size = pneighbors_end - pneighbors_start;

    for (int i = wd.number_of_members[WIB_IDX] + LANE_IDX; i < wd.total_vertices[WIB_IDX]; 
         i += WARP_SIZE) {
        
        phelper1 = ld.vertices[i].vertexid;
        phelper2 = d_b_search_int(dd->twohop_neighbors + pneighbors_start, pneighbors_size, 
                                  phelper1);

        if (phelper2 > -1) {
            ld.vertices[i].label = 0;

            // only track mem degs of candidates which pass basic degree pruning
            if(ld.vertices[i].out_mem_deg + ld.vertices[i].out_can_deg >= min_out_deg
               && ld.vertices[i].in_mem_deg + ld.vertices[i].in_can_deg >= min_in_deg){
                
                dd->lane_candidate_out_mem_degs[lane_write + lane_remaining_count] = 
                    ld.vertices[i].out_mem_deg;
                dd->lane_candidate_in_mem_degs[lane_write + lane_remaining_count] = 
                    ld.vertices[i].in_mem_deg;
                lane_remaining_count++;
            }
        }
    }
    __syncwarp();

    //  the following section combine the lane mem degs arrays into one warp array
    // scan to calculate write postion in warp arrays
    phelper2 = lane_remaining_count;
    for (int i = 1; i < WARP_SIZE; i *= 2) {
        phelper1 = __shfl_up_sync(0xFFFFFFFF, lane_remaining_count, i, WARP_SIZE);
        if (LANE_IDX >= i) {
            lane_remaining_count += phelper1;
        }
        __syncwarp();
    }
    // lane remaining count sum is scan for last lane and its value
    if (LANE_IDX == WARP_SIZE - 1) {
        wd.remaining_count[WIB_IDX] = lane_remaining_count;
    }
    // make scan exclusive
    lane_remaining_count -= phelper2;
    __syncwarp();

    // parallel write lane arrays to warp array
    for (int i = 0; i < phelper2; i++) {
        dd->candidate_out_mem_degs[(*dd->WVERTICES_SIZE * WARP_IDX) + lane_remaining_count + i] = 
            dd->lane_candidate_out_mem_degs[lane_write + i];
        dd->candidate_in_mem_degs[(*dd->WVERTICES_SIZE * WARP_IDX) + lane_remaining_count + i] = 
            dd->lane_candidate_in_mem_degs[lane_write + i];
    }
    __syncwarp();
}

// DQC - implement
__device__ void d_diameter_pruning_cv(GPU_Data* dd, Warp_Data& wd, Local_Data& ld, 
                                      int number_of_crit_adj)
{
    // int lane_write;
    // int lane_remaining_count;           // vertex iteration
    // int phelper1;                       // intersection
    // int phelper2;

    // lane_write = (*dd->WVERTICES_SIZE * WARP_IDX) + ((*dd->WVERTICES_SIZE / WARP_SIZE) * LANE_IDX);
    // lane_remaining_count = 0;

    // // remove all cands who are not within 2hops of all newly added cands
    // for (int k = wd.number_of_members[WIB_IDX] + LANE_IDX; k < wd.total_vertices[WIB_IDX]; k += WARP_SIZE) {
    //     if (dd->adjacencies[(*dd->WVERTICES_SIZE * WARP_IDX) + k] == number_of_crit_adj) {
    //         dd->lane_candidate_indegs[lane_write + lane_remaining_count++] = ld.vertices[k].indeg;
    //     }
    //     else {
    //         ld.vertices[k].label = -1;
    //     }
    // }

    // // scan to calculate write postion in warp arrays
    // phelper2 = lane_remaining_count;
    // for (int i = 1; i < WARP_SIZE; i *= 2) {
    //     phelper1 = __shfl_up_sync(0xFFFFFFFF, lane_remaining_count, i, WARP_SIZE);
    //     if (LANE_IDX >= i) {
    //         lane_remaining_count += phelper1;
    //     }
    //     __syncwarp();
    // }
    // // lane remaining count sum is scan for last lane and its value
    // if (LANE_IDX == WARP_SIZE - 1) {
    //     wd.remaining_count[WIB_IDX] = lane_remaining_count;
    // }
    // // make scan exclusive
    // lane_remaining_count -= phelper2;
    // __syncwarp();

    // // parallel write lane arrays to warp array
    // for (int i = 0; i < phelper2; i++) {
    //     dd->candidate_indegs[(*dd->WVERTICES_SIZE * WARP_IDX) + lane_remaining_count + i] = dd->lane_candidate_indegs[lane_write + i];
    // }
    // __syncwarp();
}

// returns true if invalid bounds or failed found
// DQC - implement bounds
__device__ void d_degree_pruning(GPU_Data* dd, Warp_Data& wd, Local_Data& ld)
{
    int lane_write;                 // place each lane will write in warp array
    int pvertexid;                  // helper variables
    int phelper1;
    int phelper2;
    uint64_t pneighbors_start;
    uint64_t pneighbors_end;
    uint64_t pneighbors_size;
    Vertex* read;
    Vertex* write;
    int lane_remaining_count;       // counter for lane intersection results
    int lane_removed_count;

    // TODO - add warp write variable here and in other pruning methods
    // vertices size * warp idx + (vertices size / warp size) * lane idx
    lane_write = (*dd->WVERTICES_SIZE * WARP_IDX) + ((*dd->WVERTICES_SIZE / WARP_SIZE) * LANE_IDX);

    // used for bound calculation
    d_oe_sort_int(dd->candidate_out_mem_degs + (*dd->WVERTICES_SIZE * WARP_IDX), 
                  wd.remaining_count[WIB_IDX], d_comp_int_desc);
    d_oe_sort_int(dd->candidate_in_mem_degs + (*dd->WVERTICES_SIZE * WARP_IDX), 
                  wd.remaining_count[WIB_IDX], d_comp_int_desc);

    // DQC - make it so it sets success as false if bounds fail
    // d_calculate_LU_bounds(dd, wd, ld, wd.remaining_count[WIB_IDX]);
    // if (wd.success[WIB_IDX]) {
    //     return true;
    // }

    // check for failed vertices
    __syncwarp();
    for (int k = LANE_IDX; k < wd.number_of_members[WIB_IDX] && wd.success[WIB_IDX]; 
         k += WARP_SIZE) {
        
        if (!d_vert_isextendable(ld.vertices[k], dd, wd, ld)) {
            wd.success[WIB_IDX] = false;
            break;
        }

    }
    __syncwarp();
    if (!wd.success[WIB_IDX]) {
        return;
    }

    if (LANE_IDX == 0) {
        wd.remaining_count[WIB_IDX] = 0;
        wd.removed_count[WIB_IDX] = 0;
        wd.rw_counter[WIB_IDX] = 0;
    }

    lane_remaining_count = 0;
    lane_removed_count = 0;
    
    // check for invalid candidates
    for (int i = wd.number_of_members[WIB_IDX] + LANE_IDX; i < wd.total_vertices[WIB_IDX]; 
         i += WARP_SIZE) {
        
        if (ld.vertices[i].label == 0 && d_cand_isvalid(ld.vertices[i], dd, wd, ld)) {
            dd->lane_remaining_candidates[lane_write + lane_remaining_count++] = i;
        }
        else {
            dd->lane_removed_candidates[lane_write + lane_removed_count++] = i;
        }
    }
    __syncwarp();

    // scan to calculate write postion in warp arrays
    // TODO - combine if statement with use of extra helper
    phelper2 = lane_remaining_count;
    pvertexid = lane_removed_count;
    for (int i = 1; i < WARP_SIZE; i *= 2) {
        phelper1 = __shfl_up_sync(0xFFFFFFFF, lane_remaining_count, i, WARP_SIZE);
        if (LANE_IDX >= i) {
            lane_remaining_count += phelper1;
        }
        phelper1 = __shfl_up_sync(0xFFFFFFFF, lane_removed_count, i, WARP_SIZE);
        if (LANE_IDX >= i) {
            lane_removed_count += phelper1;
        }
        __syncwarp();
    }
    // lane remaining count sum is scan for last lane and its value
    if (LANE_IDX == WARP_SIZE - 1) {
        wd.remaining_count[WIB_IDX] = lane_remaining_count;
        wd.removed_count[WIB_IDX] = lane_removed_count;
    }
    // make scan exclusive
    lane_remaining_count -= phelper2;
    lane_removed_count -= pvertexid;

    // parallel write lane arrays to warp array
    for (int i = 0; i < phelper2; i++) {
        dd->remaining_candidates[(*dd->WVERTICES_SIZE * WARP_IDX) + lane_remaining_count + i] = 
            ld.vertices[dd->lane_remaining_candidates[lane_write + i]];
    }
    for (int i = 0; i < pvertexid; i++) {
        dd->removed_candidates[(*dd->WVERTICES_SIZE * WARP_IDX) + lane_removed_count + i] = 
            ld.vertices[dd->lane_removed_candidates[lane_write + i]].vertexid;
    }
    __syncwarp();
    
    while (wd.remaining_count[WIB_IDX] > 0 && wd.removed_count[WIB_IDX] > 0) {
        
        // we alternate reading and writing remaining variables from two arrays
        if (wd.rw_counter[WIB_IDX] % 2 == 0) {
            read = dd->remaining_candidates + (*dd->WVERTICES_SIZE * WARP_IDX);
            write = ld.vertices + wd.number_of_members[WIB_IDX];
        }
        else {
            read = ld.vertices + wd.number_of_members[WIB_IDX];
            write = dd->remaining_candidates + (*dd->WVERTICES_SIZE * WARP_IDX);
        }

        // update degrees
        if (wd.remaining_count[WIB_IDX] < wd.removed_count[WIB_IDX]) {
            
            // via remaining, reset exdegs
            for (int i = LANE_IDX; i < wd.number_of_members[WIB_IDX]; i += WARP_SIZE) {
                ld.vertices[i].in_can_deg = 0;
                ld.vertices[i].out_can_deg = 0;
            }
            for (int i = LANE_IDX; i < wd.remaining_count[WIB_IDX]; i += WARP_SIZE) {
                read[i].in_can_deg = 0;
                read[i].out_can_deg = 0;
            }
            __syncwarp();

            // update exdeg based on remaining candidates, every lane should get the next vertex to intersect dynamically
            for (int i = LANE_IDX; i < wd.number_of_members[WIB_IDX]; i += WARP_SIZE) {
                
                pvertexid = ld.vertices[i].vertexid;

                pneighbors_start = dd->out_offsets[pvertexid];
                pneighbors_end = dd->out_offsets[pvertexid + 1];
                pneighbors_size = pneighbors_end - pneighbors_start;

                for (int j = 0; j < wd.remaining_count[WIB_IDX]; j++) {
                    
                    phelper1 = read[j].vertexid;
                    phelper2 = d_b_search_int(dd->out_neighbors + pneighbors_start, 
                                              pneighbors_size, phelper1);

                    if (phelper2 > -1) {
                        ld.vertices[i].out_can_deg++;
                    }
                }

                pneighbors_start = dd->in_offsets[pvertexid];
                pneighbors_end = dd->in_offsets[pvertexid + 1];
                pneighbors_size = pneighbors_end - pneighbors_start;

                for (int j = 0; j < wd.remaining_count[WIB_IDX]; j++) {
                    
                    phelper1 = read[j].vertexid;
                    phelper2 = d_b_search_int(dd->in_neighbors + pneighbors_start, 
                                              pneighbors_size, phelper1);

                    if (phelper2 > -1) {
                        ld.vertices[i].in_can_deg++;
                    }
                }
            }

            for (int i = LANE_IDX; i < wd.remaining_count[WIB_IDX]; i += WARP_SIZE) {
                
                pvertexid = read[i].vertexid;

                pneighbors_start = dd->out_offsets[pvertexid];
                pneighbors_end = dd->out_offsets[pvertexid + 1];
                pneighbors_size = pneighbors_end - pneighbors_start;

                for (int j = 0; j < wd.remaining_count[WIB_IDX]; j++) {

                    phelper1 = read[j].vertexid;
                    phelper2 = d_b_search_int(dd->out_neighbors + pneighbors_start, 
                                              pneighbors_size, phelper1);

                    if (phelper2 > -1) {
                        read[i].out_can_deg++;
                    }
                }

                pneighbors_start = dd->in_offsets[pvertexid];
                pneighbors_end = dd->in_offsets[pvertexid + 1];
                pneighbors_size = pneighbors_end - pneighbors_start;

                for (int j = 0; j < wd.remaining_count[WIB_IDX]; j++) {

                    phelper1 = read[j].vertexid;
                    phelper2 = d_b_search_int(dd->in_neighbors + pneighbors_start, 
                                              pneighbors_size, phelper1);

                    if (phelper2 > -1) {
                        read[i].in_can_deg++;
                    }
                }
            }
        }
        else {
            
            // via removed, update exdeg based on remaining candidates, again lane scheduling should be dynamic
            for (int i = LANE_IDX; i < wd.number_of_members[WIB_IDX]; i += WARP_SIZE) {
                
                pvertexid = ld.vertices[i].vertexid;

                pneighbors_start = dd->out_offsets[pvertexid];
                pneighbors_end = dd->out_offsets[pvertexid + 1];
                pneighbors_size = pneighbors_end - pneighbors_start;

                for (int j = 0; j < wd.removed_count[WIB_IDX]; j++) {
                    
                    phelper1 = dd->removed_candidates[(*dd->WVERTICES_SIZE * WARP_IDX) + j];
                    phelper2 = d_b_search_int(dd->out_neighbors + pneighbors_start, 
                                              pneighbors_size, phelper1);

                    if (phelper2 > -1) {
                        ld.vertices[i].out_can_deg--;
                    }
                }

                pneighbors_start = dd->in_offsets[pvertexid];
                pneighbors_end = dd->in_offsets[pvertexid + 1];
                pneighbors_size = pneighbors_end - pneighbors_start;

                for (int j = 0; j < wd.removed_count[WIB_IDX]; j++) {
                    
                    phelper1 = dd->removed_candidates[(*dd->WVERTICES_SIZE * WARP_IDX) + j];
                    phelper2 = d_b_search_int(dd->in_neighbors + pneighbors_start, 
                                              pneighbors_size, phelper1);

                    if (phelper2 > -1) {
                        ld.vertices[i].in_can_deg--;
                    }
                }
            }

            for (int i = LANE_IDX; i < wd.remaining_count[WIB_IDX]; i += WARP_SIZE) {
                
                pvertexid = read[i].vertexid;

                pneighbors_start = dd->out_offsets[pvertexid];
                pneighbors_end = dd->out_offsets[pvertexid + 1];
                pneighbors_size = pneighbors_end - pneighbors_start;

                for (int j = 0; j < wd.removed_count[WIB_IDX]; j++) {
                    phelper1 = dd->removed_candidates[(*dd->WVERTICES_SIZE * WARP_IDX) + j];
                    phelper2 = d_b_search_int(dd->out_neighbors + pneighbors_start, 
                                              pneighbors_size, phelper1);

                    if (phelper2 > -1) {
                        read[i].out_can_deg--;
                    }
                }

                pneighbors_start = dd->in_offsets[pvertexid];
                pneighbors_end = dd->in_offsets[pvertexid + 1];
                pneighbors_size = pneighbors_end - pneighbors_start;

                for (int j = 0; j < wd.removed_count[WIB_IDX]; j++) {
                    phelper1 = dd->removed_candidates[(*dd->WVERTICES_SIZE * WARP_IDX) + j];
                    phelper2 = d_b_search_int(dd->in_neighbors + pneighbors_start, 
                                              pneighbors_size, phelper1);

                    if (phelper2 > -1) {
                        read[i].in_can_deg--;
                    }
                }
            }
        }
        __syncwarp();

        lane_remaining_count = 0;

        for (int i = LANE_IDX; i < wd.remaining_count[WIB_IDX]; i += WARP_SIZE) {
            if (d_cand_isvalid(read[i], dd, wd, ld)) {
                dd->lane_candidate_out_mem_degs[lane_write + lane_remaining_count] = 
                    read[i].out_mem_deg;
                dd->lane_candidate_in_mem_degs[lane_write + lane_remaining_count] = 
                    read[i].in_mem_deg;
                lane_remaining_count++;
            }
        }
        __syncwarp();

        // scan to calculate write postion in warp arrays
        phelper2 = lane_remaining_count;
        for (int i = 1; i < WARP_SIZE; i *= 2) {
            phelper1 = __shfl_up_sync(0xFFFFFFFF, lane_remaining_count, i, WARP_SIZE);
            if (LANE_IDX >= i) {
                lane_remaining_count += phelper1;
            }
            __syncwarp();
        }
        // lane remaining count sum is scan for last lane and its value
        if (LANE_IDX == WARP_SIZE - 1) {
            wd.num_val_cands[WIB_IDX] = lane_remaining_count;
        }
        // make scan exclusive
        lane_remaining_count -= phelper2;

        // parallel write lane arrays to warp array
        for (int i = 0; i < phelper2; i++) {
            dd->candidate_out_mem_degs[(*dd->WVERTICES_SIZE * WARP_IDX) + lane_remaining_count + 
                i] = dd->lane_candidate_out_mem_degs[lane_write + i];
            dd->candidate_in_mem_degs[(*dd->WVERTICES_SIZE * WARP_IDX) + lane_remaining_count + 
                i] = dd->lane_candidate_in_mem_degs[lane_write + i];
        }
        __syncwarp();

        d_oe_sort_int(dd->candidate_out_mem_degs + (*dd->WVERTICES_SIZE * WARP_IDX), 
                      wd.num_val_cands[WIB_IDX], d_comp_int_desc);
        d_oe_sort_int(dd->candidate_in_mem_degs + (*dd->WVERTICES_SIZE * WARP_IDX), 
                      wd.num_val_cands[WIB_IDX], d_comp_int_desc);

        // DQC - make it so it sets success as false if bounds fail
        // d_calculate_LU_bounds(dd, wd, ld, wd.num_val_cands[WIB_IDX]);
        // if (wd.success[WIB_IDX]) {
        //     return true;
        // }

        // check for failed vertices
        for (int k = LANE_IDX; k < wd.number_of_members[WIB_IDX] && wd.success[WIB_IDX]; k += WARP_SIZE) {
            if (!d_vert_isextendable(ld.vertices[k], dd, wd, ld)) {
                wd.success[WIB_IDX] = false;
                break;
            }

        }
        __syncwarp();
        if (!wd.success[WIB_IDX]) {
            return;
        }

        lane_remaining_count = 0;
        lane_removed_count = 0;

        // check for failed candidates
        for (int i = LANE_IDX; i < wd.remaining_count[WIB_IDX]; i += WARP_SIZE) {
            if (d_cand_isvalid(read[i], dd, wd, ld)) {
                dd->lane_remaining_candidates[lane_write + lane_remaining_count++] = i;
            }
            else {
                dd->lane_removed_candidates[lane_write + lane_removed_count++] = i;
            }
        }
        __syncwarp();

        // scan to calculate write postion in warp arrays
        phelper2 = lane_remaining_count;
        pvertexid = lane_removed_count;
        for (int i = 1; i < WARP_SIZE; i *= 2) {
            phelper1 = __shfl_up_sync(0xFFFFFFFF, lane_remaining_count, i, WARP_SIZE);
            if (LANE_IDX >= i) {
                lane_remaining_count += phelper1;
            }
            phelper1 = __shfl_up_sync(0xFFFFFFFF, lane_removed_count, i, WARP_SIZE);
            if (LANE_IDX >= i) {
                lane_removed_count += phelper1;
            }
            __syncwarp();
        }
        // lane remaining count sum is scan for last lane and its value
        if (LANE_IDX == WARP_SIZE - 1) {
            wd.num_val_cands[WIB_IDX] = lane_remaining_count;
            wd.removed_count[WIB_IDX] = lane_removed_count;
        }
        // make scan exclusive
        lane_remaining_count -= phelper2;
        lane_removed_count -= pvertexid;

        // parallel write lane arrays to warp array
        for (int i = 0; i < phelper2; i++) {
            write[lane_remaining_count + i] = read[dd->lane_remaining_candidates[lane_write + i]];
        }
        // only need removed if going to be using removed to update degrees
        if (!(wd.num_val_cands[WIB_IDX] < wd.removed_count[WIB_IDX])) {
            for (int i = 0; i < pvertexid; i++) {
                dd->removed_candidates[(*dd->WVERTICES_SIZE * WARP_IDX) + lane_removed_count + i] = read[dd->lane_removed_candidates[lane_write + i]].vertexid;
            }
        }

        if (LANE_IDX == 0) {
            wd.remaining_count[WIB_IDX] = wd.num_val_cands[WIB_IDX];
            wd.rw_counter[WIB_IDX]++;
        }
    }

    // condense vertices so remaining are after members, only needs to be done if they were not written into vertices last time
    if (wd.rw_counter[WIB_IDX] % 2 == 0) {
        for (int i = LANE_IDX; i < wd.remaining_count[WIB_IDX]; i += WARP_SIZE) {
            ld.vertices[wd.number_of_members[WIB_IDX] + i] = dd->remaining_candidates[(*dd->WVERTICES_SIZE * WARP_IDX) + i];
        }
    }

    if (LANE_IDX == 0) {
        wd.total_vertices[WIB_IDX] = wd.total_vertices[WIB_IDX] - wd.number_of_candidates[WIB_IDX] + wd.remaining_count[WIB_IDX];
        wd.number_of_candidates[WIB_IDX] = wd.remaining_count[WIB_IDX];
    }
    __syncwarp();
}

// DQC - implement
__device__ void d_calculate_LU_bounds(GPU_Data* dd, Warp_Data& wd, Local_Data& ld, 
                                      int number_of_candidates)
{
    // int index;
    // int min_clq_indeg;
    // int min_indeg_exdeg;
    // int min_clq_totaldeg;
    // int sum_clq_indeg;

    // // initialize the values of the LU calculation variables to the first vertices values so they can be compared to other vertices without error
    // min_clq_indeg = ld.vertices[0].indeg;
    // min_indeg_exdeg = ld.vertices[0].exdeg;
    // min_clq_totaldeg = ld.vertices[0].indeg + ld.vertices[0].exdeg;
    // sum_clq_indeg = 0;

    // // each warp also has a copy of these variables to allow for intra-warp comparison of these variables.
    // if (LANE_IDX == 0) {
    //     wd.success[WIB_IDX] = false;

    //     wd.sum_candidate_indeg[WIB_IDX] = 0;
    //     wd.tightened_upper_bound[WIB_IDX] = 0;

    //     wd.min_clq_indeg[WIB_IDX] = ld.vertices[0].indeg;
    //     wd.min_indeg_exdeg[WIB_IDX] = ld.vertices[0].exdeg;
    //     wd.min_clq_totaldeg[WIB_IDX] = ld.vertices[0].indeg + ld.vertices[0].exdeg;
    //     wd.sum_clq_indeg[WIB_IDX] = ld.vertices[0].indeg;

    //     wd.min_ext_deg[WIB_IDX] = d_get_mindeg(wd.number_of_members[WIB_IDX] + 1, dd);
    // }
    // __syncwarp();

    // // each warp finds these values on their subsection of vertices
    // for (index = 1 + LANE_IDX; index < wd.number_of_members[WIB_IDX]; index += WARP_SIZE) {
    //     sum_clq_indeg += ld.vertices[index].indeg;

    //     if (ld.vertices[index].indeg < min_clq_indeg) {
    //         min_clq_indeg = ld.vertices[index].indeg;
    //         min_indeg_exdeg = ld.vertices[index].exdeg;
    //     }
    //     else if (ld.vertices[index].indeg == min_clq_indeg) {
    //         if (ld.vertices[index].exdeg < min_indeg_exdeg) {
    //             min_indeg_exdeg = ld.vertices[index].exdeg;
    //         }
    //     }

    //     if (ld.vertices[index].indeg + ld.vertices[index].exdeg < min_clq_totaldeg) {
    //         min_clq_totaldeg = ld.vertices[index].indeg + ld.vertices[index].exdeg;
    //     }
    // }

    // // get sum
    // for (int i = 1; i < 32; i *= 2) {
    //     sum_clq_indeg += __shfl_xor_sync(0xFFFFFFFF, sum_clq_indeg, i);
    // }
    // if (LANE_IDX == 0) {
    //     // add to shared memory sum
    //     wd.sum_clq_indeg[WIB_IDX] += sum_clq_indeg;
    // }
    // __syncwarp();

    // // CRITICAL SECTION - each lane then compares their values to the next to get a warp level value
    // for (int i = 0; i < WARP_SIZE; i++) {
    //     if (LANE_IDX == i) {
    //         if (min_clq_indeg < wd.min_clq_indeg[WIB_IDX]) {
    //             wd.min_clq_indeg[WIB_IDX] = min_clq_indeg;
    //             wd.min_indeg_exdeg[WIB_IDX] = min_indeg_exdeg;
    //         }
    //         else if (min_clq_indeg == wd.min_clq_indeg[WIB_IDX]) {
    //             if (min_indeg_exdeg < wd.min_indeg_exdeg[WIB_IDX]) {
    //                 wd.min_indeg_exdeg[WIB_IDX] = min_indeg_exdeg;
    //             }
    //         }

    //         if (min_clq_totaldeg < wd.min_clq_totaldeg[WIB_IDX]) {
    //             wd.min_clq_totaldeg[WIB_IDX] = min_clq_totaldeg;
    //         }
    //     }
    //     __syncwarp();
    // }

    // // CRITICAL SECTION - only first lane does this as there are little calculations
    // if (LANE_IDX == 0) {
    //     if (wd.min_clq_indeg[WIB_IDX] < dd->minimum_degrees[wd.number_of_members[WIB_IDX]])
    //     {
    //         // lower
    //         wd.lower_bound[WIB_IDX] = d_get_mindeg(wd.number_of_members[WIB_IDX], dd) - min_clq_indeg;

    //         while (wd.lower_bound[WIB_IDX] <= wd.min_indeg_exdeg[WIB_IDX] && wd.min_clq_indeg[WIB_IDX] + wd.lower_bound[WIB_IDX] <
    //             dd->minimum_degrees[wd.number_of_members[WIB_IDX] + wd.lower_bound[WIB_IDX]]) {
    //             wd.lower_bound[WIB_IDX]++;
    //         }

    //         if (wd.min_clq_indeg[WIB_IDX] + wd.lower_bound[WIB_IDX] < dd->minimum_degrees[wd.number_of_members[WIB_IDX] + wd.lower_bound[WIB_IDX]]) {
    //             wd.success[WIB_IDX] = true;
    //         }

    //         // upper
    //         wd.upper_bound[WIB_IDX] = floor(wd.min_clq_totaldeg[WIB_IDX] / (*(dd->minimum_degree_ratio))) + 1 - wd.number_of_members[WIB_IDX];

    //         if (wd.upper_bound[WIB_IDX] > number_of_candidates) {
    //             wd.upper_bound[WIB_IDX] = number_of_candidates;
    //         }

    //         // tighten
    //         if (wd.lower_bound[WIB_IDX] < wd.upper_bound[WIB_IDX]) {
    //             // tighten lower
    //             for (index = 0; index < wd.lower_bound[WIB_IDX]; index++) {
    //                 wd.sum_candidate_indeg[WIB_IDX] += dd->candidate_indegs[(*dd->WVERTICES_SIZE * WARP_IDX) + index];
    //             }

    //             while (index < wd.upper_bound[WIB_IDX] && wd.sum_clq_indeg[WIB_IDX] + wd.sum_candidate_indeg[WIB_IDX] < wd.number_of_members[WIB_IDX] *
    //                 dd->minimum_degrees[wd.number_of_members[WIB_IDX] + index]) {
    //                 wd.sum_candidate_indeg[WIB_IDX] += dd->candidate_indegs[(*dd->WVERTICES_SIZE * WARP_IDX) + index];
    //                 index++;
    //             }

    //             if (wd.sum_clq_indeg[WIB_IDX] + wd.sum_candidate_indeg[WIB_IDX] < wd.number_of_members[WIB_IDX] * dd->minimum_degrees[wd.number_of_members[WIB_IDX] + index]) {
    //                 wd.success[WIB_IDX] = true;
    //             }
    //             else {
    //                 wd.lower_bound[WIB_IDX] = index;

    //                 wd.tightened_upper_bound[WIB_IDX] = index;

    //                 while (index < wd.upper_bound[WIB_IDX]) {
    //                     wd.sum_candidate_indeg[WIB_IDX] += dd->candidate_indegs[(*dd->WVERTICES_SIZE * WARP_IDX) + index];

    //                     index++;

    //                     if (wd.sum_clq_indeg[WIB_IDX] + wd.sum_candidate_indeg[WIB_IDX] >= wd.number_of_members[WIB_IDX] *
    //                         dd->minimum_degrees[wd.number_of_members[WIB_IDX] + index]) {
    //                         wd.tightened_upper_bound[WIB_IDX] = index;
    //                     }
    //                 }

    //                 if (wd.upper_bound[WIB_IDX] > wd.tightened_upper_bound[WIB_IDX]) {
    //                     wd.upper_bound[WIB_IDX] = wd.tightened_upper_bound[WIB_IDX];
    //                 }

    //                 if (wd.lower_bound[WIB_IDX] > 1) {
    //                     wd.min_ext_deg[WIB_IDX] = d_get_mindeg(wd.number_of_members[WIB_IDX] + wd.lower_bound[WIB_IDX], dd);
    //                 }
    //             }
    //         }
    //     }
    //     else {
    //         wd.min_ext_deg[WIB_IDX] = d_get_mindeg(wd.number_of_members[WIB_IDX] + 1,
    //             dd);

    //         wd.upper_bound[WIB_IDX] = number_of_candidates;

    //         if (wd.number_of_members[WIB_IDX] < (*(dd->minimum_clique_size))) {
    //             wd.lower_bound[WIB_IDX] = (*(dd->minimum_clique_size)) - wd.number_of_members[WIB_IDX];
    //         }
    //         else {
    //             wd.lower_bound[WIB_IDX] = 0;
    //         }
    //     }

    //     if (wd.number_of_members[WIB_IDX] + wd.upper_bound[WIB_IDX] < (*(dd->minimum_clique_size))) {
    //         wd.success[WIB_IDX] = true;
    //     }

    //     if (wd.upper_bound[WIB_IDX] < 0 || wd.upper_bound[WIB_IDX] < wd.lower_bound[WIB_IDX]) {
    //         wd.success[WIB_IDX] = true;
    //     }
    // }
    // __syncwarp();
}

// TODO - make a write clique method
__device__ void d_check_for_clique(GPU_Data* dd, Warp_Data& wd, Local_Data& ld)
{
    uint64_t start_write;
    bool clique;
    int min_out_deg;
    int min_in_deg;

    if (wd.number_of_members[WIB_IDX] < *dd->minimum_clique_size) {
        return;
    }

    clique = true;

    min_out_deg = dd->minimum_out_degrees[wd.number_of_members[WIB_IDX]];
    min_in_deg = dd->minimum_in_degrees[wd.number_of_members[WIB_IDX]];

    for (int k = LANE_IDX; k < wd.number_of_members[WIB_IDX]; k += WARP_SIZE) {
        if (ld.vertices[k].out_mem_deg < min_out_deg || ld.vertices[k].in_mem_deg < min_in_deg) {
            clique = false;
            break;
        }
    }
    // set to false if any threads in warp do not meet degree requirement
    clique = !(__any_sync(0xFFFFFFFF, !clique));

    // if clique write to warp buffer for cliques
    if (clique) {
        start_write = (*dd->WCLIQUES_SIZE * WARP_IDX) + 
            dd->wcliques_offset[(*dd->WCLIQUES_OFFSET_SIZE * WARP_IDX) + 
            dd->wcliques_count[WARP_IDX]];

        for (int k = LANE_IDX; k < wd.number_of_members[WIB_IDX]; k += WARP_SIZE) {
            dd->wcliques_vertex[start_write + k] = ld.vertices[k].vertexid;
        }
        if (LANE_IDX == 0) {
            (dd->wcliques_count[WARP_IDX])++;

            dd->wcliques_offset[*dd->WCLIQUES_OFFSET_SIZE * WARP_IDX + 
                dd->wcliques_count[WARP_IDX]] = start_write - (*dd->WCLIQUES_SIZE * WARP_IDX) + 
                wd.number_of_members[WIB_IDX];
        }
    }
}

__device__ void d_write_to_tasks(GPU_Data* dd, Warp_Data& wd, Local_Data& ld)
{
    uint64_t start_write;

    start_write = (*dd->WTASKS_SIZE * WARP_IDX) + dd->wtasks_offset[*dd->WTASKS_OFFSET_SIZE * WARP_IDX + dd->wtasks_count[WARP_IDX]];

    for (int k = LANE_IDX; k < wd.total_vertices[WIB_IDX]; k += WARP_SIZE) {
        dd->wtasks_vertices[start_write + k] = ld.vertices[k];
        dd->wtasks_vertices[start_write + k].lvl2adj = 0;
    }
    if (LANE_IDX == 0) {
        dd->wtasks_count[WARP_IDX]++;
        dd->wtasks_offset[(*dd->WTASKS_OFFSET_SIZE * WARP_IDX) + dd->wtasks_count[WARP_IDX]] = start_write - (*dd->WTASKS_SIZE * WARP_IDX) + wd.total_vertices[WIB_IDX];
    }
}

// --- TERTIARY KENERLS ---
// searches an int array for a certain int, returns the position in the array that item was found, 
// or -1 if not found
__device__ int d_b_search_int(int* search_array, int array_size, int search_number)
{
    // ALGO - BINARY
    // TYPE - SERIAL
    // SPEED - O(log(n))
    
    int low;
    int high;
    int mid;
    int mid_value;
    int comp;

    low = 0;
    high = array_size - 1;

    while (low < high) {
        mid = (low + high) / 2;
        mid_value = search_array[mid];
        comp = (mid_value < search_number);

        low = low + comp * (mid + 1 - low);
        high = high - !comp * (high - mid);
    }

    // Now low == high, check if it's the search_number
    return (search_array[low] == search_number) ? low : -1;
}

// consider using merge
__device__ void d_oe_sort_vert(Vertex* target, int size, int (*func)(Vertex&, Vertex&))
{
    // ALGO - ODD/EVEN
    // TYPE - PARALLEL
    // SPEED - O(n^2)

    Vertex vertex1;
    Vertex vertex2;

    for (int i = 0; i < size; i++) {
        for (int j = (i % 2) + (LANE_IDX * 2); j < size - 1; j += (WARP_SIZE * 2)) {
            vertex1 = target[j];
            vertex2 = target[j + 1];

            if (func(vertex1, vertex2) == 1) {
                target[j] = vertex2;
                target[j + 1] = vertex1;
            }
        }
        __syncwarp();
    }
}

__device__ void d_oe_sort_int(int* target, int size, int (*func)(int, int))
{
    // ALGO - ODD/EVEN
    // TYPE - PARALLEL
    // SPEED - O(n^2)

    int num1;
    int num2;

    for (int i = 0; i < size; i++) {
        for (int j = (i % 2) + (LANE_IDX * 2); j < size - 1; j += (WARP_SIZE * 2)) {
            num1 = target[j];
            num2 = target[j + 1];

            if (func(num1, num2) == 1) {
                target[j] = num2;
                target[j + 1] = num1;
            }
        }
        __syncwarp();
    }
}

// --- DEBUG KERNELS ---
// __device__ void d_print_vertices(Vertex* vertices, int size)
// {
//     printf("\nOffsets:\n0 %i\nVertex:\n", size);
//     for (int i = 0; i < size; i++) {
//         printf("%i ", vertices[i].vertexid);
//     }
//     printf("\nLabel:\n");
//     for (int i = 0; i < size; i++) {
//         printf("%i ", vertices[i].label);
//     }
//     printf("\nIndeg:\n");
//     for (int i = 0; i < size; i++) {
//         printf("%i ", vertices[i].indeg);
//     }
//     printf("\nExdeg:\n");
//     for (int i = 0; i < size; i++) {
//         printf("%i ", vertices[i].exdeg);
//     }
//     printf("\nLvl2adj:\n");
//     for (int i = 0; i < size; i++) {
//         printf("%i ", vertices[i].lvl2adj);
//     }
//     printf("\n");
// }