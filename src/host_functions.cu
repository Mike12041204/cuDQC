#include "hip/hip_runtime.h"
#include "../inc/common.h"
#include "../inc/host_functions.h"
#include "../inc/host_debug.h"
#include "../inc/device_kernels.h"

// --- PRIMARY FUNCTIONS ---
// initializes minimum degrees array 
void calculate_minimum_degrees(CPU_Graph& hg, int*& minimum_degrees, double minimum_degree_ratio)
{
    minimum_degrees = new int[hg.number_of_vertices + 1];
    minimum_degrees[0] = 0;
    for (int i = 1; i <= hg.number_of_vertices; i++) {
        minimum_degrees[i] = ceil(minimum_degree_ratio * (i - 1));
    }
}

void search(CPU_Graph& hg, ofstream& temp_results, ofstream& output_file, DS_Sizes& dss, int* minimum_degrees, double minimum_degree_ratio, int minimum_clique_size) 
{
    // DATA STRUCTURES
    CPU_Data hd;
    CPU_Cliques hc;
    GPU_Data h_dd;
    GPU_Data* dd;


    // HANDLE MEMORY
    allocate_memory(hd, h_dd, hc, hg, dss, minimum_degrees, minimum_degree_ratio, minimum_clique_size);
    hipDeviceSynchronize();
    chkerr(hipMalloc((void**)&dd, sizeof(GPU_Data)));
    chkerr(hipMemcpy(dd, &h_dd, sizeof(GPU_Data), hipMemcpyHostToDevice));
    hipDeviceSynchronize();



    // TIME
    auto start = chrono::high_resolution_clock::now();



    // INITIALIZE TASKS
    if(grank == 0){
        cout << ">:INITIALIZING TASKS" << endl;
    }
    initialize_tasks(hg, hd, minimum_degrees, minimum_clique_size);



    // DEBUG
    if (DEBUG_TOGGLE) {
        mvs = (*(hd.tasks1_offset + (*hd.tasks1_count)));
        if ((*(hd.tasks1_offset + (*hd.tasks1_count))) > dss.wvertices_size) {
            cout << "!!! VERTICES SIZE ERROR !!!" << endl;
            return;
        }
        h_print_Data_Sizes(hd, hc, output_file);
    }



    // CPU EXPANSION
    // cpu levels is multiplied by two to ensure that data ends up in tasks1, this allows us to always copy tasks1 without worry like before hybrid cpu approach
    // cpu expand must be called atleast one time to handle first round cover pruning as the gpu code cannot do this
    for (int i = 0; i < CPU_LEVELS + 1 && !(*hd.maximal_expansion); i++) {
        h_expand_level(hg, hd, hc, dss, minimum_degrees, minimum_degree_ratio, minimum_clique_size);
    
        // if cliques is more than threshold dump
        if (hc.cliques_offset[(*hc.cliques_count)] > dss.cliques_dump) {
            flush_cliques(hc, temp_results);
        }



        // DEBUG
        if (DEBUG_TOGGLE) {
            h_print_Data_Sizes(hd, hc, output_file);
        }
    }

    flush_cliques(hc, temp_results);



    // TRANSFER TO GPU
    move_to_gpu(hd, h_dd, dss);
    hipDeviceSynchronize();



    // TODO - use cuTS distributed loop
    // EXPAND LEVEL
    if(grank == 0){
        cout << ">:BEGINNING EXPANSION" << endl;
    }
    while (!(*hd.maximal_expansion))
    {
        (*(hd.maximal_expansion)) = true;
        chkerr(hipMemset(h_dd.current_task, 0, sizeof(int)));
        hipDeviceSynchronize();

        // expand all tasks in 'tasks' array, each warp will write to their respective warp tasks buffer in global memory
        d_expand_level<<<NUM_OF_BLOCKS, BLOCK_SIZE>>>(dd);
        hipDeviceSynchronize();



        // DEBUG
        if (DEBUG_TOGGLE) {
            if (print_Warp_Data_Sizes_Every(h_dd, 1, output_file, dss)) { break; }
        }



        // consolidate all the warp tasks/cliques buffers into the next global tasks array, buffer, and cliques
        transfer_buffers<<<NUM_OF_BLOCKS, BLOCK_SIZE>>>(dd);
        hipDeviceSynchronize();



        // determine whether maximal expansion has been accomplished
        uint64_t current_level, write_count, buffer_count;
        // TODO - do we still need to copy current level from the GPU or can we just have a counter on the CPU or handle it on the GPU
        chkerr(hipMemcpy(&current_level, h_dd.current_level, sizeof(uint64_t), hipMemcpyDeviceToHost));
        chkerr(hipMemcpy(&buffer_count, h_dd.buffer_count, sizeof(uint64_t), hipMemcpyDeviceToHost));
        chkerr(hipMemcpy(&write_count, h_dd.tasks1_count, sizeof(uint64_t), hipMemcpyDeviceToHost));

        if (write_count > 0 || buffer_count > 0) {
            (*(hd.maximal_expansion)) = false;
        }

        // TODO - what number of tasks is enough to split



        chkerr(hipMemset(h_dd.wtasks_count, 0, sizeof(uint64_t) * NUMBER_OF_WARPS));
        chkerr(hipMemset(h_dd.wcliques_count, 0, sizeof(uint64_t) * NUMBER_OF_WARPS));
        if (write_count < dss.expand_threshold && buffer_count > 0) {
            // if not enough tasks were generated when expanding the previous level to fill the next tasks array the program will attempt to fill the tasks array by popping tasks from the buffer
            fill_from_buffer<<<NUM_OF_BLOCKS, BLOCK_SIZE>>>(dd);
            hipDeviceSynchronize();
        }
        current_level++;
        chkerr(hipMemcpy(h_dd.current_level, &current_level, sizeof(uint64_t), hipMemcpyHostToDevice));




        // determine whether cliques has exceeded defined threshold, if so dump them to a file
        uint64_t cliques_size, cliques_count;
        chkerr(hipMemcpy(&cliques_count, h_dd.cliques_count, sizeof(uint64_t), hipMemcpyDeviceToHost));
        chkerr(hipMemcpy(&cliques_size, h_dd.cliques_offset + cliques_count, sizeof(uint64_t), hipMemcpyDeviceToHost));
        hipDeviceSynchronize();

        // if cliques is more than threshold dump
        if (cliques_size > dss.cliques_dump) {
            dump_cliques(hc, h_dd, temp_results, dss);
        }



        // DEBUG
        if (DEBUG_TOGGLE) {
            if (print_Data_Sizes_Every(h_dd, 1, output_file, dss)) { break; }
        }
    }



    // TIME
    auto stop = chrono::high_resolution_clock::now();
    auto duration = chrono::duration_cast<chrono::milliseconds>(stop - start);
    MPI_Barrier(MPI_COMM_WORLD);
    if(grank == 0){
        cout << "--->:ENUMERATION TIME: " << duration.count() << " ms" << endl;
    }



    dump_cliques(hc, h_dd, temp_results, dss);

    free_memory(hd, h_dd, hc);
    chkerr(hipFree(dd));
}

// allocates memory for the data structures on the host and device   
void allocate_memory(CPU_Data& hd, GPU_Data& h_dd, CPU_Cliques& hc, CPU_Graph& hg, DS_Sizes& dss, int* minimum_degrees, double minimum_degree_ratio, int minimum_clique_size)
{
    // GPU GRAPH
    chkerr(hipMalloc((void**)&h_dd.number_of_vertices, sizeof(int)));
    chkerr(hipMalloc((void**)&h_dd.number_of_edges, sizeof(int)));
    chkerr(hipMalloc((void**)&h_dd.onehop_neighbors, sizeof(int) * hg.number_of_edges));
    chkerr(hipMalloc((void**)&h_dd.onehop_offsets, sizeof(uint64_t) * (hg.number_of_vertices + 1)));
    chkerr(hipMalloc((void**)&h_dd.twohop_neighbors, sizeof(int) * hg.number_of_lvl2adj));
    chkerr(hipMalloc((void**)&h_dd.twohop_offsets, sizeof(uint64_t) * (hg.number_of_vertices + 1)));

    chkerr(hipMemcpy(h_dd.number_of_vertices, &(hg.number_of_vertices), sizeof(int), hipMemcpyHostToDevice));
    chkerr(hipMemcpy(h_dd.number_of_edges, &(hg.number_of_edges), sizeof(int), hipMemcpyHostToDevice));
    chkerr(hipMemcpy(h_dd.onehop_neighbors, hg.onehop_neighbors, sizeof(int) * hg.number_of_edges, hipMemcpyHostToDevice));
    chkerr(hipMemcpy(h_dd.onehop_offsets, hg.onehop_offsets, sizeof(uint64_t) * (hg.number_of_vertices + 1), hipMemcpyHostToDevice));
    chkerr(hipMemcpy(h_dd.twohop_neighbors, hg.twohop_neighbors, sizeof(int) * hg.number_of_lvl2adj, hipMemcpyHostToDevice));
    chkerr(hipMemcpy(h_dd.twohop_offsets, hg.twohop_offsets, sizeof(uint64_t) * (hg.number_of_vertices + 1), hipMemcpyHostToDevice));
    // CPU DATA
    hd.tasks1_count = new uint64_t;
    hd.tasks1_offset = new uint64_t[dss.expand_threshold + 1];
    hd.tasks1_vertices = new Vertex[dss.tasks_size];

    hd.tasks1_offset[0] = 0;
    (*(hd.tasks1_count)) = 0;

    hd.tasks2_count = new uint64_t;
    hd.tasks2_offset = new uint64_t[dss.expand_threshold + 1];
    hd.tasks2_vertices = new Vertex[dss.tasks_size];

    hd.tasks2_offset[0] = 0;
    (*(hd.tasks2_count)) = 0;

    hd.buffer_count = new uint64_t;
    hd.buffer_offset = new uint64_t[dss.buffer_offset_size];
    hd.buffer_vertices = new Vertex[dss.buffer_size];

    hd.buffer_offset[0] = 0;
    (*(hd.buffer_count)) = 0;

    hd.current_level = new uint64_t;
    hd.maximal_expansion = new bool;
    hd.dumping_cliques = new bool;

    (*hd.current_level) = 0;
    (*hd.maximal_expansion) = false;
    (*hd.dumping_cliques) = false;

    hd.vertex_order_map = new int[hg.number_of_vertices];
    hd.remaining_candidates = new int[hg.number_of_vertices];
    hd.removed_candidates = new int[hg.number_of_vertices];
    hd.remaining_count = new int;
    hd.removed_count = new int;
    hd.candidate_indegs = new int[hg.number_of_vertices];

    memset(hd.vertex_order_map, -1, sizeof(int) * hg.number_of_vertices);

    // GPU DATA
    chkerr(hipMalloc((void**)&h_dd.current_level, sizeof(uint64_t)));

    chkerr(hipMalloc((void**)&h_dd.tasks1_count, sizeof(uint64_t)));
    chkerr(hipMalloc((void**)&h_dd.tasks1_offset, sizeof(uint64_t) * (dss.expand_threshold + 1)));
    chkerr(hipMalloc((void**)&h_dd.tasks1_vertices, sizeof(Vertex) * dss.tasks_size));
    chkerr(hipMemset(h_dd.tasks1_offset, 0, sizeof(uint64_t)));
    chkerr(hipMemset(h_dd.tasks1_count, 0, sizeof(uint64_t)));

    chkerr(hipMalloc((void**)&h_dd.buffer_count, sizeof(uint64_t)));
    chkerr(hipMalloc((void**)&h_dd.buffer_offset, sizeof(uint64_t) * dss.buffer_offset_size));
    chkerr(hipMalloc((void**)&h_dd.buffer_vertices, sizeof(Vertex) * dss.buffer_size));
    chkerr(hipMemset(h_dd.buffer_offset, 0, sizeof(uint64_t)));
    chkerr(hipMemset(h_dd.buffer_count, 0, sizeof(uint64_t)));

    chkerr(hipMalloc((void**)&h_dd.wtasks_count, sizeof(uint64_t) * NUMBER_OF_WARPS));
    chkerr(hipMalloc((void**)&h_dd.wtasks_offset, (sizeof(uint64_t) * dss.wtasks_offset_size) * NUMBER_OF_WARPS));
    chkerr(hipMalloc((void**)&h_dd.wtasks_vertices, (sizeof(Vertex) * dss.wtasks_size) * NUMBER_OF_WARPS));
    chkerr(hipMemset(h_dd.wtasks_offset, 0, (sizeof(uint64_t) * dss.wtasks_offset_size) * NUMBER_OF_WARPS));
    chkerr(hipMemset(h_dd.wtasks_count, 0, sizeof(uint64_t) * NUMBER_OF_WARPS));

    chkerr(hipMalloc((void**)&h_dd.global_vertices, (sizeof(Vertex) * dss.wvertices_size) * NUMBER_OF_WARPS));

    chkerr(hipMalloc((void**)&h_dd.removed_candidates, (sizeof(int) * dss.wvertices_size) * NUMBER_OF_WARPS));
    chkerr(hipMalloc((void**)&h_dd.lane_removed_candidates, (sizeof(int) * dss.wvertices_size) * NUMBER_OF_WARPS));
    chkerr(hipMalloc((void**)&h_dd.remaining_candidates, (sizeof(Vertex) * dss.wvertices_size) * NUMBER_OF_WARPS));
    chkerr(hipMalloc((void**)&h_dd.lane_remaining_candidates, (sizeof(int) * dss.wvertices_size) * NUMBER_OF_WARPS));

    chkerr(hipMalloc((void**)&h_dd.candidate_indegs, (sizeof(int) * dss.wvertices_size) * NUMBER_OF_WARPS));
    chkerr(hipMalloc((void**)&h_dd.lane_candidate_indegs, (sizeof(int) * dss.wvertices_size) * NUMBER_OF_WARPS));

    chkerr(hipMalloc((void**)&h_dd.adjacencies, (sizeof(int) * dss.wvertices_size) * NUMBER_OF_WARPS));
    chkerr(hipMalloc((void**)&h_dd.minimum_degree_ratio, sizeof(double)));
    chkerr(hipMalloc((void**)&h_dd.minimum_degrees, sizeof(int) * (hg.number_of_vertices + 1)));
    chkerr(hipMalloc((void**)&h_dd.minimum_clique_size, sizeof(int)));
    chkerr(hipMemcpy(h_dd.minimum_degree_ratio, &minimum_degree_ratio, sizeof(double), hipMemcpyHostToDevice));
    chkerr(hipMemcpy(h_dd.minimum_degrees, minimum_degrees, sizeof(int) * (hg.number_of_vertices + 1), hipMemcpyHostToDevice));
    chkerr(hipMemcpy(h_dd.minimum_clique_size, &minimum_clique_size, sizeof(int), hipMemcpyHostToDevice));
    chkerr(hipMalloc((void**)&h_dd.total_tasks, sizeof(int)));

    chkerr(hipMemset(h_dd.total_tasks, 0, sizeof(int)));
    // CPU CLIQUES
    hc.cliques_count = new uint64_t;
    hc.cliques_vertex = new int[dss.cliques_size];
    hc.cliques_offset = new uint64_t[dss.cliques_offset_size];

    hc.cliques_offset[0] = 0;
    (*(hc.cliques_count)) = 0;
    // GPU CLIQUES
    chkerr(hipMalloc((void**)&h_dd.cliques_count, sizeof(uint64_t)));
    chkerr(hipMalloc((void**)&h_dd.cliques_vertex, sizeof(int) * dss.cliques_size));
    chkerr(hipMalloc((void**)&h_dd.cliques_offset, sizeof(uint64_t) * dss.cliques_offset_size));

    chkerr(hipMemset(h_dd.cliques_offset, 0, sizeof(uint64_t)));
    chkerr(hipMemset(h_dd.cliques_count, 0, sizeof(uint64_t)));

    chkerr(hipMalloc((void**)&h_dd.wcliques_count, sizeof(uint64_t) * NUMBER_OF_WARPS));
    chkerr(hipMalloc((void**)&h_dd.wcliques_offset, (sizeof(uint64_t) * dss.wcliques_offset_size) * NUMBER_OF_WARPS));
    chkerr(hipMalloc((void**)&h_dd.wcliques_vertex, (sizeof(int) * dss.wcliques_size) * NUMBER_OF_WARPS));

    chkerr(hipMemset(h_dd.wcliques_offset, 0, (sizeof(uint64_t) * dss.wcliques_offset_size) * NUMBER_OF_WARPS));
    chkerr(hipMemset(h_dd.wcliques_count, 0, sizeof(uint64_t) * NUMBER_OF_WARPS));

    chkerr(hipMalloc((void**)&h_dd.total_cliques, sizeof(int)));

    chkerr(hipMemset(h_dd.total_cliques, 0, sizeof(int)));

    chkerr(hipMalloc((void**)&h_dd.buffer_offset_start, sizeof(uint64_t)));
    chkerr(hipMalloc((void**)&h_dd.buffer_start, sizeof(uint64_t)));
    chkerr(hipMalloc((void**)&h_dd.cliques_offset_start, sizeof(uint64_t)));
    chkerr(hipMalloc((void**)&h_dd.cliques_start, sizeof(uint64_t)));

    // task scheduling
    chkerr(hipMalloc((void**)&h_dd.current_task, sizeof(int)));
    chkerr(hipMalloc((void**)&h_dd.tasks_size, sizeof(uint64_t)));
    chkerr(hipMalloc((void**)&h_dd.tasks_per_warp, sizeof(uint64_t)));
    chkerr(hipMalloc((void**)&h_dd.buffer_size, sizeof(uint64_t)));
    chkerr(hipMalloc((void**)&h_dd.buffer_offset_size, sizeof(uint64_t)));
    chkerr(hipMalloc((void**)&h_dd.cliques_size, sizeof(uint64_t)));
    chkerr(hipMalloc((void**)&h_dd.cliques_offset_size, sizeof(uint64_t)));
    chkerr(hipMalloc((void**)&h_dd.cliques_percent, sizeof(uint64_t)));
    chkerr(hipMalloc((void**)&h_dd.wcliques_size, sizeof(uint64_t)));
    chkerr(hipMalloc((void**)&h_dd.wcliques_offset_size, sizeof(uint64_t)));
    chkerr(hipMalloc((void**)&h_dd.wtasks_size, sizeof(uint64_t)));
    chkerr(hipMalloc((void**)&h_dd.wtasks_offset_size, sizeof(uint64_t)));
    chkerr(hipMalloc((void**)&h_dd.wvertices_size, sizeof(uint64_t)));
    chkerr(hipMalloc((void**)&h_dd.expand_threshold, sizeof(uint64_t)));
    chkerr(hipMalloc((void**)&h_dd.cliques_dump, sizeof(uint64_t)));

    chkerr(hipMemcpy(h_dd.tasks_size, &dss.tasks_size, sizeof(uint64_t), hipMemcpyHostToDevice));
    chkerr(hipMemcpy(h_dd.tasks_per_warp, &dss.tasks_per_warp, sizeof(uint64_t), hipMemcpyHostToDevice));
    chkerr(hipMemcpy(h_dd.buffer_size, &dss.buffer_size, sizeof(uint64_t), hipMemcpyHostToDevice));
    chkerr(hipMemcpy(h_dd.buffer_offset_size, &dss.buffer_offset_size, sizeof(uint64_t), hipMemcpyHostToDevice));
    chkerr(hipMemcpy(h_dd.cliques_size, &dss.cliques_size, sizeof(uint64_t), hipMemcpyHostToDevice));
    chkerr(hipMemcpy(h_dd.cliques_offset_size, &dss.cliques_offset_size, sizeof(uint64_t), hipMemcpyHostToDevice));
    chkerr(hipMemcpy(h_dd.cliques_percent, &dss.cliques_percent, sizeof(uint64_t), hipMemcpyHostToDevice));
    chkerr(hipMemcpy(h_dd.wcliques_size, &dss.wcliques_size, sizeof(uint64_t), hipMemcpyHostToDevice));
    chkerr(hipMemcpy(h_dd.wcliques_offset_size, &dss.wcliques_offset_size, sizeof(uint64_t), hipMemcpyHostToDevice));
    chkerr(hipMemcpy(h_dd.wtasks_size, &dss.wtasks_size, sizeof(uint64_t), hipMemcpyHostToDevice));
    chkerr(hipMemcpy(h_dd.wtasks_offset_size, &dss.wtasks_offset_size, sizeof(uint64_t), hipMemcpyHostToDevice));
    chkerr(hipMemcpy(h_dd.wvertices_size, &dss.wvertices_size, sizeof(uint64_t), hipMemcpyHostToDevice));
    chkerr(hipMemcpy(h_dd.expand_threshold, &dss.expand_threshold, sizeof(uint64_t), hipMemcpyHostToDevice));
    chkerr(hipMemcpy(h_dd.cliques_dump, &dss.cliques_dump, sizeof(uint64_t), hipMemcpyHostToDevice));
}

// processes 0th level of expansion
void initialize_tasks(CPU_Graph& hg, CPU_Data& hd, int* minimum_degrees, int minimum_clique_size)
{
    // intersection
    int pvertexid;
    uint64_t pneighbors_start;
    uint64_t pneighbors_end;
    int phelper1;

    // cover pruning
    int maximum_degree;
    int maximum_degree_index;

    // vertices information
    int total_vertices;
    int number_of_candidates;
    Vertex* vertices;



    (*hd.remaining_count) = 0;
    (*hd.removed_count) = 0;

    // initialize vertices
    total_vertices = hg.number_of_vertices;
    vertices = new Vertex[total_vertices];
    number_of_candidates = total_vertices;
    for (int i = 0; i < total_vertices; i++) {
        vertices[i].vertexid = i;
        vertices[i].indeg = 0;
        vertices[i].exdeg = hg.onehop_offsets[i + 1] - hg.onehop_offsets[i];
        vertices[i].lvl2adj = hg.twohop_offsets[i + 1] - hg.twohop_offsets[i];
        if (vertices[i].exdeg >= minimum_degrees[minimum_clique_size] && vertices[i].lvl2adj >= minimum_clique_size - 1) {
            vertices[i].label = 0;
            hd.remaining_candidates[(*hd.remaining_count)++] = i;
        }
        else {
            vertices[i].label = -1;
            hd.removed_candidates[(*hd.removed_count)++] = i;
        }
    }

    

    // DEGREE-BASED PRUNING
    // update while half of vertices have been removed
    while ((*hd.remaining_count) < number_of_candidates / 2) {
        number_of_candidates = (*hd.remaining_count);
        
        for (int i = 0; i < number_of_candidates; i++) {
            vertices[hd.remaining_candidates[i]].exdeg = 0;
        }

        for (int i = 0; i < number_of_candidates; i++) {
            // in 0th level id is same as position in vertices as all vertices are in vertices, see last block
            pvertexid = hd.remaining_candidates[i];
            pneighbors_start = hg.onehop_offsets[pvertexid];
            pneighbors_end = hg.onehop_offsets[pvertexid + 1];
            for (int j = pneighbors_start; j < pneighbors_end; j++) {
                phelper1 = hg.onehop_neighbors[j];
                if (vertices[phelper1].label == 0) {
                    vertices[phelper1].exdeg++;
                }
            }
        }

        (*hd.remaining_count) = 0;
        (*hd.removed_count) = 0;

        // remove more vertices based on updated degrees
        for (int i = 0; i < number_of_candidates; i++) {
            phelper1 = hd.remaining_candidates[i];
            if (vertices[phelper1].exdeg >= minimum_degrees[minimum_clique_size]) {
                hd.remaining_candidates[(*hd.remaining_count)++] = phelper1;
            }
            else {
                vertices[phelper1].label = -1;
                hd.removed_candidates[(*hd.removed_count)++] = phelper1;
            }
        }
    }
    number_of_candidates = (*hd.remaining_count);

    // update degrees based on last round of removed vertices
    int removed_start = 0;
    while((*hd.removed_count) > removed_start) {
        pvertexid = hd.removed_candidates[removed_start];
        pneighbors_start = hg.onehop_offsets[pvertexid];
        pneighbors_end = hg.onehop_offsets[pvertexid + 1];

        for (int j = pneighbors_start; j < pneighbors_end; j++) {
            phelper1 = hg.onehop_neighbors[j];

            if (vertices[phelper1].label == 0) {
                vertices[phelper1].exdeg--;

                if (vertices[phelper1].exdeg < minimum_degrees[minimum_clique_size]) {
                    vertices[phelper1].label = -1;
                    number_of_candidates--;
                    hd.removed_candidates[(*hd.removed_count)++] = phelper1;
                }
            }
        }
        removed_start++;
    }


    
    // FIRST ROUND COVER PRUNING
    // find cover vertex
    maximum_degree = 0;
    maximum_degree_index = 0;
    for (int i = 0; i < total_vertices; i++) {
        if (vertices[i].label == 0) {
            if (vertices[i].exdeg > maximum_degree) {
                maximum_degree = vertices[i].exdeg;
                maximum_degree_index = i;
            }
        }
    }
    vertices[maximum_degree_index].label = 3;

    // find all covered vertices
    pneighbors_start = hg.onehop_offsets[maximum_degree_index];
    pneighbors_end = hg.onehop_offsets[maximum_degree_index + 1];
    for (int i = pneighbors_start; i < pneighbors_end; i++) {
        pvertexid = hg.onehop_neighbors[i];
        if (vertices[pvertexid].label == 0) {
            vertices[pvertexid].label = 2;
        }
    }

    // sort enumeration order before writing to tasks
    qsort(vertices, total_vertices, sizeof(Vertex), h_comp_vert_Q);
    total_vertices = number_of_candidates;



    // WRITE TO TASKS
    if (total_vertices > 0)
    {
        for (int j = 0; j < total_vertices; j++) {
            hd.tasks1_vertices[j].vertexid = vertices[j].vertexid;
            hd.tasks1_vertices[j].label = vertices[j].label;
            hd.tasks1_vertices[j].indeg = vertices[j].indeg;
            hd.tasks1_vertices[j].exdeg = vertices[j].exdeg;
            hd.tasks1_vertices[j].lvl2adj = 0;
        }
        (*(hd.tasks1_count))++;
        hd.tasks1_offset[(*(hd.tasks1_count))] = total_vertices;
    }

    delete vertices;
}

void h_expand_level(CPU_Graph& hg, CPU_Data& hd, CPU_Cliques& hc, DS_Sizes& dss, int* minimum_degrees, double minimum_degree_ratio, int minimum_clique_size)
{
    // initiate the variables containing the location of the read and write task vectors, done in an alternating, odd-even manner like the c-intersection of cuTS
    uint64_t* read_count;
    uint64_t* read_offsets;
    Vertex* read_vertices;
    uint64_t* write_count;
    uint64_t* write_offsets;
    Vertex* write_vertices;

    // old vertices information
    uint64_t start;
    uint64_t end;
    int tot_vert;
    int num_mem;
    int num_cand;
    int expansions;
    int number_of_covered;

    // new vertices information
    Vertex* vertices;
    int number_of_members;
    int number_of_candidates;
    int total_vertices;

    // calculate lower-upper bounds
    int min_ext_deg;
    int lower_bound;
    int upper_bound;

    int method_return;
    int index;



    if ((*hd.current_level) % 2 == 0) {
        read_count = hd.tasks1_count;
        read_offsets = hd.tasks1_offset;
        read_vertices = hd.tasks1_vertices;
        write_count = hd.tasks2_count;
        write_offsets = hd.tasks2_offset;
        write_vertices = hd.tasks2_vertices;
    }
    else {
        read_count = hd.tasks2_count;
        read_offsets = hd.tasks2_offset;
        read_vertices = hd.tasks2_vertices;
        write_count = hd.tasks1_count;
        write_offsets = hd.tasks1_offset;
        write_vertices = hd.tasks1_vertices;
    }
    *write_count = 0;
    write_offsets[0] = 0;

    // set to false later if task is generated indicating non-maximal expansion
    (*hd.maximal_expansion) = true;



    // CURRENT LEVEL
    for (int i = 0; i < *read_count; i++)
    {
        // get information of vertices being handled within tasks
        start = read_offsets[i];
        end = read_offsets[i + 1];
        tot_vert = end - start;
        num_mem = 0;
        for (uint64_t j = start; j < end; j++) {
            if (read_vertices[j].label != 1) {
                break;
            }
            num_mem++;
        }
        number_of_covered = 0;
        for (uint64_t j = start + num_mem; j < end; j++) {
            if (read_vertices[j].label != 2) {
                break;
            }
            number_of_covered++;
        }
        num_cand = tot_vert - num_mem;
        expansions = num_cand;



        // LOOKAHEAD PRUNING
        method_return = h_lookahead_pruning(hg, hc, hd, read_vertices, tot_vert, num_mem, num_cand, start, minimum_degrees);
        if (method_return) {
            continue;
        }



        // NEXT LEVEL
        for (int j = number_of_covered; j < expansions; j++) {



            // REMOVE ONE VERTEX
            if (j != number_of_covered) {
                method_return = h_remove_one_vertex(hg, hd, read_vertices, tot_vert, num_cand, num_mem, start, minimum_degrees, minimum_clique_size);
                if (method_return) {
                    break;
                }
            }



            // NEW VERTICES
            vertices = new Vertex[tot_vert];
            number_of_members = num_mem;
            number_of_candidates = num_cand;
            total_vertices = tot_vert;
            for (index = 0; index < number_of_members; index++) {
                vertices[index] = read_vertices[start + index];
            }
            vertices[number_of_members] = read_vertices[start + total_vertices - 1];
            for (; index < total_vertices - 1; index++) {
                vertices[index + 1] = read_vertices[start + index];
            }

            if (number_of_covered > 0) {
                // set all covered vertices from previous level as candidates
                for (int j = num_mem + 1; j <= num_mem + number_of_covered; j++) {
                    vertices[j].label = 0;
                }
            }



            // ADD ONE VERTEX
            method_return = h_add_one_vertex(hg, hd, vertices, total_vertices, number_of_candidates, number_of_members, upper_bound, lower_bound, min_ext_deg, minimum_degrees, minimum_degree_ratio, minimum_clique_size);

            // if vertex in x found as not extendable, check if current set is clique and continue to next iteration
            if (method_return == 1) {
                if (number_of_members >= minimum_clique_size) {
                    h_check_for_clique(hc, vertices, number_of_members, minimum_degrees);
                }

                delete vertices;
                continue;
            }



            // CRITICAL VERTEX PRUNING
            method_return = h_critical_vertex_pruning(hg, hd, vertices, total_vertices, number_of_candidates, number_of_members, upper_bound, lower_bound, min_ext_deg, minimum_degrees, minimum_degree_ratio, minimum_clique_size);

            // if critical fail continue onto next iteration
            if (method_return == 2) {
                delete vertices;
                continue;
            }



            // CHECK FOR CLIQUE
            // all processes will do this, to prevent duplicates only process 0 will save cpu results
            if (grank == 0 && number_of_members >= minimum_clique_size) {
                h_check_for_clique(hc, vertices, number_of_members, minimum_degrees);
            }

            // if vertex in x found as not extendable, check if current set is clique and continue to next iteration
            if (method_return == 1) {
                delete vertices;
                continue;
            }



            // WRITE TO TASKS
            //sort vertices so that lowest degree vertices are first in enumeration order before writing to tasks
            qsort(vertices, total_vertices, sizeof(Vertex), h_comp_vert_Q);

            if (number_of_candidates > 0) {
                h_write_to_tasks(hd, vertices, total_vertices, write_vertices, write_offsets, write_count);
            }



            delete vertices;
        }
    }



    // FILL TASKS FROM BUFFER
    // if last CPU round copy enough tasks for GPU expansion
    if ((*hd.current_level) == CPU_LEVELS && CPU_EXPAND_THRESHOLD < dss.expand_threshold && (*hd.buffer_count) > 0) {
        h_fill_from_buffer(hd, write_vertices, write_offsets, write_count, dss.expand_threshold);
    }
    // if not enough generated to fully populate fill from buffer
    if (*write_count < CPU_EXPAND_THRESHOLD && (*hd.buffer_count) > 0){
        h_fill_from_buffer(hd, write_vertices, write_offsets, write_count, CPU_EXPAND_THRESHOLD);
    }

    (*hd.current_level)++;
}

// TODO - distribute work amongst processes in more intelligent manner 
void move_to_gpu(CPU_Data& hd, GPU_Data& h_dd, DS_Sizes& dss)
{
    uint64_t* tasks_count;
    uint64_t* tasks_offset;
    Vertex* tasks_vertices;

    uint64_t block_size;
    uint64_t block_start;

    uint64_t offset_start;



    // split tasks
    // get proper read location for level
    if(CPU_LEVELS % 2 == 1){
        tasks_count = hd.tasks1_count;
        tasks_offset = hd.tasks1_offset;
        tasks_vertices = hd.tasks1_vertices;
    }
    else{
        tasks_count = hd.tasks2_count;
        tasks_offset = hd.tasks2_offset;
        tasks_vertices = hd.tasks2_vertices;
    }

    // get work size for tasks
    block_size = *tasks_count / NUMBER_OF_PROCESSESS;
    block_start = block_size * grank;
    if(grank == NUMBER_OF_PROCESSESS - 1){
        block_size += *tasks_count % NUMBER_OF_PROCESSESS;
    }

    // rearange tasks
    memmove(tasks_count, &block_size, sizeof(uint64_t));
    memmove(tasks_offset, tasks_offset + block_start, sizeof(uint64_t) * (block_size + 1));
    memmove(tasks_vertices, tasks_vertices + tasks_offset[0], sizeof(Vertex) * (tasks_offset[block_size] - tasks_offset[0]));

    // revalue tasks
    offset_start = tasks_offset[0];
    for(int i = 0; i <= block_size; i++){
        tasks_offset[i] -= offset_start;
    }

    // get work size for buffer
    block_size = *hd.buffer_count / NUMBER_OF_PROCESSESS;
    block_start = block_size * grank;
    if(grank == NUMBER_OF_PROCESSESS - 1){
        block_size += *hd.buffer_count % NUMBER_OF_PROCESSESS;
    }

    // rearange buffer
    memmove(hd.buffer_count, &block_size, sizeof(uint64_t));
    memmove(hd.buffer_offset, hd.buffer_offset + block_start, sizeof(uint64_t) * (block_size + 1));
    memmove(hd.buffer_vertices, hd.buffer_vertices + hd.buffer_offset[0], sizeof(Vertex) * (hd.buffer_offset[block_size] - hd.buffer_offset[0]));

    // revalue buffer
    offset_start = hd.buffer_offset[0];
    for(int i = 0; i <= block_size; i++){
        hd.buffer_offset[i] -= offset_start;
    }

    // condense tasks
    h_fill_from_buffer(hd, tasks_vertices, tasks_offset, tasks_count, dss.expand_threshold);

    // TODO - only copy whats needed
    // move to GPU
    chkerr(hipMemcpy(h_dd.tasks1_count, hd.tasks1_count, sizeof(uint64_t), hipMemcpyHostToDevice));
    chkerr(hipMemcpy(h_dd.tasks1_offset, hd.tasks1_offset, (dss.expand_threshold + 1) * sizeof(uint64_t), hipMemcpyHostToDevice));
    chkerr(hipMemcpy(h_dd.tasks1_vertices, hd.tasks1_vertices, (dss.tasks_size) * sizeof(Vertex), hipMemcpyHostToDevice));

    chkerr(hipMemcpy(h_dd.buffer_count, hd.buffer_count, sizeof(uint64_t), hipMemcpyHostToDevice));
    chkerr(hipMemcpy(h_dd.buffer_offset, hd.buffer_offset, (dss.buffer_offset_size) * sizeof(uint64_t), hipMemcpyHostToDevice));
    chkerr(hipMemcpy(h_dd.buffer_vertices, hd.buffer_vertices, (dss.buffer_size) * sizeof(int), hipMemcpyHostToDevice));

    chkerr(hipMemcpy(h_dd.current_level, hd.current_level, sizeof(uint64_t), hipMemcpyHostToDevice));
}

void dump_cliques(CPU_Cliques& hc, GPU_Data& h_dd, ofstream& temp_results, DS_Sizes& dss)
{
    // gpu cliques to cpu cliques
    chkerr(hipMemcpy(hc.cliques_count, h_dd.cliques_count, sizeof(uint64_t), hipMemcpyDeviceToHost));
    chkerr(hipMemcpy(hc.cliques_offset, h_dd.cliques_offset, sizeof(uint64_t) * dss.cliques_offset_size, hipMemcpyDeviceToHost));
    chkerr(hipMemcpy(hc.cliques_vertex, h_dd.cliques_vertex, sizeof(int) * dss.cliques_size, hipMemcpyDeviceToHost));
    hipDeviceSynchronize();

    // DEBUG
    //print_CPU_Cliques(hc);

    flush_cliques(hc, temp_results);

    hipMemset(h_dd.cliques_count, 0, sizeof(uint64_t));
}

void flush_cliques(CPU_Cliques& hc, ofstream& temp_results) 
{
    for (int i = 0; i < ((*hc.cliques_count)); i++) {
        uint64_t start = hc.cliques_offset[i];
        uint64_t end = hc.cliques_offset[i + 1];
        temp_results << end - start << " ";
        for (uint64_t j = start; j < end; j++) {
            temp_results << hc.cliques_vertex[j] << " ";
        }
        temp_results << "\n";
    }
    ((*hc.cliques_count)) = 0;
}

void free_memory(CPU_Data& hd, GPU_Data& h_dd, CPU_Cliques& hc)
{
    // GPU GRAPH
    chkerr(hipFree(h_dd.number_of_vertices));
    chkerr(hipFree(h_dd.number_of_edges));
    chkerr(hipFree(h_dd.onehop_neighbors));
    chkerr(hipFree(h_dd.onehop_offsets));
    chkerr(hipFree(h_dd.twohop_neighbors));
    chkerr(hipFree(h_dd.twohop_offsets));

    // CPU DATA
    delete hd.tasks1_count;
    delete hd.tasks1_offset;
    delete hd.tasks1_vertices;

    delete hd.tasks2_count;
    delete hd.tasks2_offset;
    delete hd.tasks2_vertices;

    delete hd.buffer_count;
    delete hd.buffer_offset;
    delete hd.buffer_vertices;

    delete hd.current_level;
    delete hd.maximal_expansion;
    delete hd.dumping_cliques;

    delete hd.vertex_order_map;
    delete hd.remaining_candidates;
    delete hd.remaining_count;
    delete hd.removed_candidates;
    delete hd.removed_count;
    delete hd.candidate_indegs;

    // GPU DATA
    chkerr(hipFree(h_dd.current_level));

    chkerr(hipFree(h_dd.tasks1_count));
    chkerr(hipFree(h_dd.tasks1_offset));
    chkerr(hipFree(h_dd.tasks1_vertices));

    chkerr(hipFree(h_dd.buffer_count));
    chkerr(hipFree(h_dd.buffer_offset));
    chkerr(hipFree(h_dd.buffer_vertices));

    chkerr(hipFree(h_dd.wtasks_count));
    chkerr(hipFree(h_dd.wtasks_offset));
    chkerr(hipFree(h_dd.wtasks_vertices));

    chkerr(hipFree(h_dd.global_vertices));

    chkerr(hipFree(h_dd.remaining_candidates));
    chkerr(hipFree(h_dd.lane_remaining_candidates));

    chkerr(hipFree(h_dd.removed_candidates));
    chkerr(hipFree(h_dd.lane_removed_candidates));

    chkerr(hipFree(h_dd.candidate_indegs));
    chkerr(hipFree(h_dd.lane_candidate_indegs));

    chkerr(hipFree(h_dd.adjacencies));

    chkerr(hipFree(h_dd.minimum_degree_ratio));
    chkerr(hipFree(h_dd.minimum_degrees));
    chkerr(hipFree(h_dd.minimum_clique_size));

    chkerr(hipFree(h_dd.total_tasks));

    // CPU CLIQUES
    delete hc.cliques_count;
    delete hc.cliques_vertex;
    delete hc.cliques_offset;

    // GPU CLIQUES
    chkerr(hipFree(h_dd.cliques_count));
    chkerr(hipFree(h_dd.cliques_vertex));
    chkerr(hipFree(h_dd.cliques_offset));

    chkerr(hipFree(h_dd.wcliques_count));
    chkerr(hipFree(h_dd.wcliques_vertex));
    chkerr(hipFree(h_dd.wcliques_offset));

    chkerr(hipFree(h_dd.buffer_offset_start));
    chkerr(hipFree(h_dd.buffer_start));
    chkerr(hipFree(h_dd.cliques_offset_start));
    chkerr(hipFree(h_dd.cliques_start));

    // tasks scheduling
    chkerr(hipFree(h_dd.current_task));
}

// --- SECONDARY EXPNASION FUNCTIONS ---
// returns 1 if lookahead was a success, else 0
int h_lookahead_pruning(CPU_Graph& hg, CPU_Cliques& hc, CPU_Data& hd, Vertex* read_vertices, int tot_vert, int num_mem, int num_cand, uint64_t start, int* minimum_degrees)
{
    // intersection
    int pvertexid;
    uint64_t pneighbors_start;
    uint64_t pneighbors_end;
    int phelper1;


    // check if members meet degree requirement, dont need to check 2hop adj as diameter pruning guarentees all members will be within 2hops of eveything
    for (int i = 0; i < num_mem; i++) {
        if (read_vertices[start + i].indeg + read_vertices[start + i].exdeg < minimum_degrees[tot_vert]) {
            return 0;
        }
    }

    // initialize vertex order map
    for (int i = num_mem; i < tot_vert; i++) {
        hd.vertex_order_map[read_vertices[start + i].vertexid] = i;
    }

    // update lvl2adj to candidates for all vertices
    for (int i = num_mem; i < tot_vert; i++) {
        pvertexid = read_vertices[start + i].vertexid;
        pneighbors_start = hg.twohop_offsets[pvertexid];
        pneighbors_end = hg.twohop_offsets[pvertexid + 1];
        for (int j = pneighbors_start; j < pneighbors_end; j++) {
            phelper1 = hd.vertex_order_map[hg.twohop_neighbors[j]];

            if (phelper1 >= num_mem) {
                read_vertices[start + phelper1].lvl2adj++;
            }
        }
    }

    // reset vertex order map
    for (int i = num_mem; i < tot_vert; i++) {
        hd.vertex_order_map[read_vertices[start + i].vertexid] = -1;
    }

    // check for lookahead
    for (int j = num_mem; j < tot_vert; j++) {
        if (read_vertices[start + j].lvl2adj < num_cand - 1 || read_vertices[start + j].indeg + read_vertices[start + j].exdeg < minimum_degrees[tot_vert]) {
            return 0;
        }
    }

    // write to cliques
    uint64_t start_write = hc.cliques_offset[(*hc.cliques_count)];
    for (int j = 0; j < tot_vert; j++) {
        hc.cliques_vertex[start_write + j] = read_vertices[start + j].vertexid;
    }
    (*hc.cliques_count)++;
    hc.cliques_offset[(*hc.cliques_count)] = start_write + tot_vert;

    return 1;
}

// returns 1 is failed found or not enough vertices, else 0
int h_remove_one_vertex(CPU_Graph& hg, CPU_Data& hd, Vertex* read_vertices, int& tot_vert, int& num_cand, int& num_mem, uint64_t start, int* minimum_degrees, int minimum_clique_size)
{
    // intersection
    int pvertexid;
    uint64_t pneighbors_start;
    uint64_t pneighbors_end;
    int phelper1;

    // helper variables
    int mindeg;
    bool failed_found;



    mindeg = h_get_mindeg(num_mem, minimum_degrees, minimum_clique_size);

    // remove one vertex
    num_cand--;
    tot_vert--;

    // initialize vertex order map
    for (int i = 0; i < tot_vert; i++) {
        hd.vertex_order_map[read_vertices[start + i].vertexid] = i;
    }

    failed_found = false;

    // update info of vertices connected to removed cand
    pvertexid = read_vertices[start + tot_vert].vertexid;
    pneighbors_start = hg.onehop_offsets[pvertexid];
    pneighbors_end = hg.onehop_offsets[pvertexid + 1];
    for (int i = pneighbors_start; i < pneighbors_end; i++) {
        phelper1 = hd.vertex_order_map[hg.onehop_neighbors[i]];

        if (phelper1 > -1) {
            read_vertices[start + phelper1].exdeg--;

            if (phelper1 < num_mem && read_vertices[start + phelper1].indeg + read_vertices[start + phelper1].exdeg < mindeg) {
                failed_found = true;
                break;
            }
        }
    }

    // reset vertex order map
    for (int i = 0; i < tot_vert; i++) {
        hd.vertex_order_map[read_vertices[start + i].vertexid] = -1;
    }

    if (failed_found) {
        return 1;
    }

    return 0;
}

// returns 1 if failed found or invalid bound, 0 otherwise
int h_add_one_vertex(CPU_Graph& hg, CPU_Data& hd, Vertex* vertices, int& total_vertices, int& number_of_candidates, int& number_of_members, int& upper_bound, int& lower_bound, int& min_ext_deg, int* minimum_degrees, double minimum_degree_ratio, int minimum_clique_size)
{
    // helper variables
    bool method_return;

    // intersection
    int pvertexid;
    uint64_t pneighbors_start;
    uint64_t pneighbors_end;
    int pneighbors_count;
    int phelper1;



    // Ah_dd ONE VERTEX
    pvertexid = vertices[number_of_members].vertexid;

    vertices[number_of_members].label = 1;
    number_of_members++;
    number_of_candidates--;

    // initialize vertex order map
    for (int i = 0; i < total_vertices; i++) {
        hd.vertex_order_map[vertices[i].vertexid] = i;
    }

    pneighbors_start = hg.onehop_offsets[pvertexid];
    pneighbors_end = hg.onehop_offsets[pvertexid + 1];
    pneighbors_count = pneighbors_end - pneighbors_start;
    for (int i = 0; i < pneighbors_count; i++) {
        phelper1 = hd.vertex_order_map[hg.onehop_neighbors[pneighbors_start + i]];

        if (phelper1 > -1) {
            vertices[phelper1].indeg++;
            vertices[phelper1].exdeg--;
        }
    }



    // DIAMETER PRUNING
    h_diameter_pruning(hg, hd, vertices, pvertexid, total_vertices, number_of_candidates, number_of_members);



    // DEGREE-BASED PRUNING
    method_return = h_degree_pruning(hg, hd, vertices, total_vertices, number_of_candidates, number_of_members, upper_bound, lower_bound, min_ext_deg, minimum_degrees, minimum_degree_ratio, minimum_clique_size);

    for (int i = 0; i < hg.number_of_vertices; i++) {
        hd.vertex_order_map[i] = -1;
    }

    // if vertex in x found as not extendable, check if current set is clique and continue to next iteration
    if (method_return) {
        return 1;
    }

    return 0;
}

// returns 2 if too many vertices pruned or a critical vertex fail, returns 1 if failed found or invalid bounds, else 0
int h_critical_vertex_pruning(CPU_Graph& hg, CPU_Data& hd, Vertex* vertices, int& total_vertices, int& number_of_candidates, int& number_of_members, int& upper_bound, int& lower_bound, int& min_ext_deg, int* minimum_degrees, double minimum_degree_ratio, int minimum_clique_size)
{
    // intersection
    int pvertexid;
    uint64_t pneighbors_start;
    uint64_t pneighbors_end;
    int phelper1;

    bool critical_fail;
    int number_of_crit_adj;
    int* adj_counters;

    bool method_return;



    // initialize vertex order map
    for (int i = 0; i < total_vertices; i++) {
        hd.vertex_order_map[vertices[i].vertexid] = i;
    }

    // CRITICAL VERTEX PRUNING
    // adj_counter[0] = 10, means that the vertex at position 0 in new_vertices has 10 critical vertices neighbors within 2 hops
    adj_counters = new int[total_vertices];
    memset(adj_counters, 0, sizeof(int) * total_vertices);

    // iterate through all vertices in clique
    for (int k = 0; k < number_of_members; k++)
    {
        // if they are a critical vertex
        if (vertices[k].indeg + vertices[k].exdeg == minimum_degrees[number_of_members + lower_bound] && vertices[k].exdeg > 0) {
            pvertexid = vertices[k].vertexid;

            // iterate through all neighbors
            pneighbors_start = hg.onehop_offsets[pvertexid];
            pneighbors_end = hg.onehop_offsets[pvertexid + 1];
            for (uint64_t l = pneighbors_start; l < pneighbors_end; l++) {
                phelper1 = hd.vertex_order_map[hg.onehop_neighbors[l]];

                // if neighbor is cand
                if (phelper1 >= number_of_members) {
                    vertices[phelper1].label = 4;
                }
            }
        }
    }



    // reset vertex order map
    for (int i = 0; i < total_vertices; i++) {
        hd.vertex_order_map[vertices[i].vertexid] = -1;
    }

    // sort vertices so that critical vertex adjacent candidates are immediately after vertices within the clique
    qsort(vertices + number_of_members, number_of_candidates, sizeof(Vertex), h_comp_vert_cv);

    // calculate number of critical adjacent vertices
    number_of_crit_adj = 0;
    for (int i = number_of_members; i < total_vertices; i++) {
        if (vertices[i].label == 4) {
            number_of_crit_adj++;
        }
        else {
            break;
        }
    }



    // if there were any neighbors of critical vertices
    if (number_of_crit_adj > 0)
    {
        // initialize vertex order map
        for (int i = 0; i < total_vertices; i++) {
            hd.vertex_order_map[vertices[i].vertexid] = i;
        }

        // iterate through all neighbors
        for (int i = number_of_members; i < number_of_members + number_of_crit_adj; i++) {
            pvertexid = vertices[i].vertexid;

            // update 1hop adj
            pneighbors_start = hg.onehop_offsets[pvertexid];
            pneighbors_end = hg.onehop_offsets[pvertexid + 1];
            for (uint64_t k = pneighbors_start; k < pneighbors_end; k++) {
                phelper1 = hd.vertex_order_map[hg.onehop_neighbors[k]];

                if (phelper1 > -1) {
                    vertices[phelper1].indeg++;
                    vertices[phelper1].exdeg--;
                }
            }

            // track 2hop adj
            pneighbors_start = hg.twohop_offsets[pvertexid];
            pneighbors_end = hg.twohop_offsets[pvertexid + 1];
            for (uint64_t k = pneighbors_start; k < pneighbors_end; k++) {
                phelper1 = hd.vertex_order_map[hg.twohop_neighbors[k]];

                if (phelper1 > -1) {
                    adj_counters[phelper1]++;
                }
            }
        }

        critical_fail = false;

        // all vertices within the clique must be within 2hops of the newly ah_dded critical vertex adj vertices
        for (int k = 0; k < number_of_members; k++) {
            if (adj_counters[k] != number_of_crit_adj) {
                critical_fail = true;
            }
        }

        if (critical_fail) {
            // reset vertex order map
            for (int i = 0; i < total_vertices; i++) {
                hd.vertex_order_map[vertices[i].vertexid] = -1;
            }
            delete adj_counters;
            return 2;
        }

        // all critical adj vertices must all be within 2 hops of each other
        for (int k = number_of_members; k < number_of_members + number_of_crit_adj; k++) {
            if (adj_counters[k] < number_of_crit_adj - 1) {
                critical_fail = true;
            }
        }

        if (critical_fail) {
            // reset vertex order map
            for (int i = 0; i < total_vertices; i++) {
                hd.vertex_order_map[vertices[i].vertexid] = -1;
            }
            delete adj_counters;
            return 2;
        }

        // no failed vertices found so ah_dd all critical vertex adj candidates to clique
        for (int k = number_of_members; k < number_of_members + number_of_crit_adj; k++) {
            vertices[k].label = 1;
        }
        number_of_members += number_of_crit_adj;
        number_of_candidates -= number_of_crit_adj;
    }



    // DIAMTER PRUNING
    (*hd.remaining_count) = 0;

    // remove all cands who are not within 2hops of all newly ah_dded cands
    for (int k = number_of_members; k < total_vertices; k++) {
        if (adj_counters[k] == number_of_crit_adj) {
            hd.candidate_indegs[(*hd.remaining_count)++] = vertices[k].indeg;
        }
        else {
            vertices[k].label = -1;
        }
    }

    

    // DEGREE-BASED PRUNING
    method_return = h_degree_pruning(hg, hd, vertices, total_vertices, number_of_candidates, number_of_members, upper_bound, lower_bound, min_ext_deg, minimum_degrees, minimum_degree_ratio, minimum_clique_size);

    // reset vertex order map
    for (int i = 0; i < total_vertices; i++) {
        hd.vertex_order_map[vertices[i].vertexid] = -1;
    }

    delete adj_counters;

    // if vertex in x found as not extendable, check if current set is clique and continue to next iteration
    if (method_return) {
        return 1;
    }

    return 0;
}

void h_diameter_pruning(CPU_Graph& hg, CPU_Data& hd, Vertex* vertices, int pvertexid, int& total_vertices, int& number_of_candidates, int number_of_members)
{
    // intersection
    uint64_t pneighbors_start;
    uint64_t pneighbors_end;
    int phelper1;

    (*hd.remaining_count) = 0;

    for (int i = number_of_members; i < total_vertices; i++) {
        vertices[i].label = -1;
    }

    pneighbors_start = hg.twohop_offsets[pvertexid];
    pneighbors_end = hg.twohop_offsets[pvertexid + 1];
    for (int i = pneighbors_start; i < pneighbors_end; i++) {
        phelper1 = hd.vertex_order_map[hg.twohop_neighbors[i]];

        if (phelper1 >= number_of_members) {
            vertices[phelper1].label = 0;
            hd.candidate_indegs[(*hd.remaining_count)++] = vertices[phelper1].indeg;
        }
    }
}

// returns true is invalid bounds calculated or a failed vertex was found, else false
bool h_degree_pruning(CPU_Graph& hg, CPU_Data& hd, Vertex* vertices, int& total_vertices, int& number_of_candidates, int number_of_members, int& upper_bound, int& lower_bound, int& min_ext_deg, int* minimum_degrees, double minimum_degree_ratio, int minimum_clique_size)
{
    // intersection
    int pvertexid;
    uint64_t pneighbors_start;
    uint64_t pneighbors_end;
    int phelper1;

    // helper variables
    int num_val_cands;

    qsort(hd.candidate_indegs, (*hd.remaining_count), sizeof(int), h_comp_int_desc);

    // if invalid bounds found while calculating lower and upper bounds
    if (h_calculate_LU_bounds(hd, upper_bound, lower_bound, min_ext_deg, vertices, number_of_members, (*hd.remaining_count), minimum_degrees, minimum_degree_ratio, minimum_clique_size)) {
        return true;
    }

    // check for failed vertices
    for (int k = 0; k < number_of_members; k++) {
        if (!h_vert_isextendable(vertices[k], number_of_members, upper_bound, lower_bound, min_ext_deg, minimum_degrees, minimum_clique_size)) {
            return true;
        }
    }

    (*hd.remaining_count) = 0;
    (*hd.removed_count) = 0;

    // check for invalid candidates
    for (int i = number_of_members; i < total_vertices; i++) {
        if (vertices[i].label == 0 && h_cand_isvalid(vertices[i], number_of_members, upper_bound, lower_bound, min_ext_deg, minimum_degrees, minimum_clique_size)) {
            hd.remaining_candidates[(*hd.remaining_count)++] = i;
        }
        else {
            hd.removed_candidates[(*hd.removed_count)++] = i;
        }
    }

    while ((*hd.remaining_count) > 0 && (*hd.removed_count) > 0) {
        // update degrees
        if ((*hd.remaining_count) < (*hd.removed_count)) {
            // reset exdegs
            for (int i = 0; i < total_vertices; i++) {
                vertices[i].exdeg = 0;
            }

            for (int i = 0; i < (*hd.remaining_count); i++) {
                pvertexid = vertices[hd.remaining_candidates[i]].vertexid;
                pneighbors_start = hg.onehop_offsets[pvertexid];
                pneighbors_end = hg.onehop_offsets[pvertexid + 1];
                for (int j = pneighbors_start; j < pneighbors_end; j++) {
                    phelper1 = hd.vertex_order_map[hg.onehop_neighbors[j]];

                    if (phelper1 > -1) {
                        vertices[phelper1].exdeg++;
                    }
                }
            }
        }
        else {
            for (int i = 0; i < (*hd.removed_count); i++) {
                pvertexid = vertices[hd.removed_candidates[i]].vertexid;
                pneighbors_start = hg.onehop_offsets[pvertexid];
                pneighbors_end = hg.onehop_offsets[pvertexid + 1];
                for (int j = pneighbors_start; j < pneighbors_end; j++) {
                    phelper1 = hd.vertex_order_map[hg.onehop_neighbors[j]];

                    if (phelper1 > -1) {
                        vertices[phelper1].exdeg--;
                    }
                }
            }
        }

        num_val_cands = 0;

        for (int k = 0; k < (*hd.remaining_count); k++) {
            if (h_cand_isvalid(vertices[hd.remaining_candidates[k]], number_of_members, upper_bound, lower_bound, min_ext_deg, minimum_degrees, minimum_clique_size)) {
                hd.candidate_indegs[num_val_cands++] = vertices[hd.remaining_candidates[k]].indeg;
            }
        }

        qsort(hd.candidate_indegs, num_val_cands, sizeof(int), h_comp_int_desc);

        // if invalid bounds found while calculating lower and upper bounds
        if (h_calculate_LU_bounds(hd, upper_bound, lower_bound, min_ext_deg, vertices, number_of_members, num_val_cands, minimum_degrees, minimum_degree_ratio, minimum_clique_size)) {
            return true;
        }

        // check for failed vertices
        for (int k = 0; k < number_of_members; k++) {
            if (!h_vert_isextendable(vertices[k], number_of_members, upper_bound, lower_bound, min_ext_deg, minimum_degrees, minimum_clique_size)) {
                return true;
            }
        }

        num_val_cands = 0;
        (*hd.removed_count) = 0;

        // check for invalid candidates
        for (int k = 0; k < (*hd.remaining_count); k++) {
            if (h_cand_isvalid(vertices[hd.remaining_candidates[k]], number_of_members, upper_bound, lower_bound, min_ext_deg, minimum_degrees, minimum_clique_size)) {
                hd.remaining_candidates[num_val_cands++] = hd.remaining_candidates[k];
            }
            else {
                hd.removed_candidates[(*hd.removed_count)++] = hd.remaining_candidates[k];
            }
        }

        (*hd.remaining_count) = num_val_cands;
    }

    for (int i = 0; i < (*hd.remaining_count); i++) {
        vertices[number_of_members + i] = vertices[hd.remaining_candidates[i]];
    }

    total_vertices = total_vertices - number_of_candidates + (*hd.remaining_count);
    number_of_candidates = (*hd.remaining_count);

    return false;
}

bool h_calculate_LU_bounds(CPU_Data& hd, int& upper_bound, int& lower_bound, int& min_ext_deg, Vertex* vertices, int number_of_members, int number_of_candidates, int* minimum_degrees, double minimum_degree_ratio, int minimum_clique_size)
{
    bool invalid_bounds = false;
    int index;

    int sum_candidate_indeg = 0;
    int tightened_upper_bound = 0;

    int min_clq_indeg = vertices[0].indeg;
    int min_indeg_exdeg = vertices[0].exdeg;
    int min_clq_totaldeg = vertices[0].indeg + vertices[0].exdeg;
    int sum_clq_indeg = vertices[0].indeg;

    for (index = 1; index < number_of_members; index++) {
        sum_clq_indeg += vertices[index].indeg;

        if (vertices[index].indeg < min_clq_indeg) {
            min_clq_indeg = vertices[index].indeg;
            min_indeg_exdeg = vertices[index].exdeg;
        }
        else if (vertices[index].indeg == min_clq_indeg) {
            if (vertices[index].exdeg < min_indeg_exdeg) {
                min_indeg_exdeg = vertices[index].exdeg;
            }
        }

        if (vertices[index].indeg + vertices[index].exdeg < min_clq_totaldeg) {
            min_clq_totaldeg = vertices[index].indeg + vertices[index].exdeg;
        }
    }

    min_ext_deg = h_get_mindeg(number_of_members + 1, minimum_degrees, minimum_clique_size);

    if (min_clq_indeg < minimum_degrees[number_of_members])
    {
        // lower
        lower_bound = h_get_mindeg(number_of_members, minimum_degrees, minimum_clique_size) - min_clq_indeg;

        while (lower_bound <= min_indeg_exdeg && min_clq_indeg + lower_bound < minimum_degrees[number_of_members + lower_bound]) {
            lower_bound++;
        }

        if (min_clq_indeg + lower_bound < minimum_degrees[number_of_members + lower_bound]) {
            lower_bound = number_of_candidates + 1;
            invalid_bounds = true;
        }

        // upper
        upper_bound = floor(min_clq_totaldeg / minimum_degree_ratio) + 1 - number_of_members;

        if (upper_bound > number_of_candidates) {
            upper_bound = number_of_candidates;
        }

        // tighten
        if (lower_bound < upper_bound) {
            // tighten lower
            for (index = 0; index < lower_bound; index++) {
                sum_candidate_indeg += hd.candidate_indegs[index];
            }

            while (index < upper_bound && sum_clq_indeg + sum_candidate_indeg < number_of_members * minimum_degrees[number_of_members + index]) {
                sum_candidate_indeg += hd.candidate_indegs[index];
                index++;
            }

            if (sum_clq_indeg + sum_candidate_indeg < number_of_members * minimum_degrees[number_of_members + index]) {
                lower_bound = upper_bound + 1;
                invalid_bounds = true;
            }
            else {
                lower_bound = index;

                tightened_upper_bound = index;

                while (index < upper_bound) {
                    sum_candidate_indeg += hd.candidate_indegs[index];

                    index++;

                    if (sum_clq_indeg + sum_candidate_indeg >= number_of_members * minimum_degrees[number_of_members + index]) {
                        tightened_upper_bound = index;
                    }
                }

                if (upper_bound > tightened_upper_bound) {
                    upper_bound = tightened_upper_bound;
                }

                if (lower_bound > 1) {
                    min_ext_deg = h_get_mindeg(number_of_members + lower_bound, minimum_degrees, minimum_clique_size);
                }
            }
        }
    }
    else {
        upper_bound = number_of_candidates;

        if (number_of_members < minimum_clique_size) {
            lower_bound = minimum_clique_size - number_of_members;
        }
        else {
            lower_bound = 0;
        }
    }

    if (number_of_members + upper_bound < minimum_clique_size) {
        invalid_bounds = true;
    }

    if (upper_bound < 0 || upper_bound < lower_bound) {
        invalid_bounds = true;
    }

    return invalid_bounds;
}

void h_check_for_clique(CPU_Cliques& hc, Vertex* vertices, int number_of_members, int* minimum_degrees)
{
    bool clique = true;

    int degree_requirement = minimum_degrees[number_of_members];
    for (int k = 0; k < number_of_members; k++) {
        if (vertices[k].indeg < degree_requirement) {
            clique = false;
            break;
        }
    }

    // if clique write to cliques array
    if (clique) {
        uint64_t start_write = hc.cliques_offset[(*hc.cliques_count)];
        for (int k = 0; k < number_of_members; k++) {
            hc.cliques_vertex[start_write + k] = vertices[k].vertexid;
        }
        (*hc.cliques_count)++;
        hc.cliques_offset[(*hc.cliques_count)] = start_write + number_of_members;
    }
}

void h_write_to_tasks(CPU_Data& hd, Vertex* vertices, int total_vertices, Vertex* write_vertices, uint64_t* write_offsets, uint64_t* write_count)
{
    (*hd.maximal_expansion) = false;

    if ((*write_count) < CPU_EXPAND_THRESHOLD) {
        uint64_t start_write = write_offsets[*write_count];

        for (int k = 0; k < total_vertices; k++) {
            write_vertices[start_write + k].vertexid = vertices[k].vertexid;
            write_vertices[start_write + k].label = vertices[k].label;
            write_vertices[start_write + k].indeg = vertices[k].indeg;
            write_vertices[start_write + k].exdeg = vertices[k].exdeg;
            write_vertices[start_write + k].lvl2adj = 0;
        }
        (*write_count)++;
        write_offsets[*write_count] = start_write + total_vertices;
    }
    else {
        uint64_t start_write = hd.buffer_offset[(*hd.buffer_count)];

        for (int k = 0; k < total_vertices; k++) {
            hd.buffer_vertices[start_write + k].vertexid = vertices[k].vertexid;
            hd.buffer_vertices[start_write + k].label = vertices[k].label;
            hd.buffer_vertices[start_write + k].indeg = vertices[k].indeg;
            hd.buffer_vertices[start_write + k].exdeg = vertices[k].exdeg;
            hd.buffer_vertices[start_write + k].lvl2adj = 0;
        }
        (*hd.buffer_count)++;
        hd.buffer_offset[(*hd.buffer_count)] = start_write + total_vertices;
    }
}

void h_fill_from_buffer(CPU_Data& hd, Vertex* write_vertices, uint64_t* write_offsets, uint64_t* write_count, int threshold)
{
    // read from end of buffer, write to end of tasks, decrement buffer
    (*hd.maximal_expansion) = false;

    // get read and write locations
    int write_amount = ((*hd.buffer_count) >= (threshold - *write_count)) ? threshold - *write_count : (*hd.buffer_count);
    uint64_t start_buffer = hd.buffer_offset[(*hd.buffer_count) - write_amount];
    uint64_t end_buffer = hd.buffer_offset[(*hd.buffer_count)];
    uint64_t size_buffer = end_buffer - start_buffer;
    uint64_t start_write = write_offsets[*write_count];

    // copy tasks data from end of buffer to end of tasks
    memcpy(&write_vertices[start_write], &hd.buffer_vertices[start_buffer], sizeof(Vertex) * size_buffer);

    // handle offsets
    for (int j = 1; j <= write_amount; j++) {
        write_offsets[*write_count + j] = start_write + (hd.buffer_offset[(*hd.buffer_count) - write_amount + j] - start_buffer);
    }

    // update counts
    (*write_count) += write_amount;
    (*hd.buffer_count) -= write_amount;
}